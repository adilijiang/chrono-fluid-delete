#include "hip/hip_runtime.h"
#include "custom_cutil_math.h"
#include "SPHCudaUtils.h"
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include "collideSphereSphere.cuh"
#include "SDKCollisionSystem.cuh"
#include "FlexibleBodies.cuh"
#include "printToFile.cuh"
#include <string.h>
#include <stdio.h>
#include <math.h>
#include <sstream>
#include <fstream>
using namespace std;
//#####################################################################################
#define B_SIZE 128
//#####################################################################################
__constant__ int numAllMarkersD;
__constant__ real_ dTD;
__constant__ real_ solid_SPH_massD;
__constant__ int2 updatePortionD;
__constant__ real3 cMinD;
__constant__ real3 cMaxD;
__constant__ int2 portionD;
__constant__ int flagD;
__constant__ int numRigidBodiesD;
__constant__ int startRigidMarkersD;
__constant__ int startFlexMarkersD;
__constant__ int numRigid_SphMarkersD;
__constant__ int numFlex_SphMarkersD;

int maxblock = 65535;
//--------------------------------------------------------------------------------------------------------------------------------
__device__ __host__ void Applied_Force(real_* f_a, real_ x, real_ L, real3 F)
{
	real_ S[4];

	shape_fun(S, x, L);

	f_a[0]  = F.x*S[0];
	f_a[1]  = F.y*S[0];
	f_a[2]  = F.z*S[0];
	f_a[3]  = F.x*S[1];
	f_a[4]  = F.y*S[1];
	f_a[5]  = F.z*S[1];
	f_a[6]  = F.x*S[2];
	f_a[7]  = F.y*S[2];
	f_a[8]  = F.z*S[2];
	f_a[9]  = F.x*S[3];
	f_a[10] = F.y*S[3];
	f_a[11] = F.z*S[3];
}
//--------------------------------------------------------------------------------------------------------------------------------
__device__ __host__ inline real3 Calc_ANCF_Point_Pos(
		real3 * ANCF_NodesD,
		real3 * ANCF_SlopesD,
		int indexOfClosestNode,
		real_ s,
		real_ l){
	real_ S[4];
	shape_fun(S, s, l);


	real3 r;
	real3 ni = ANCF_NodesD[indexOfClosestNode];
	real3 si = ANCF_SlopesD[indexOfClosestNode];
	real3 nj = ANCF_NodesD[indexOfClosestNode + 1];
	real3 sj = ANCF_SlopesD[indexOfClosestNode + 1];

	r.x = S[0]*ni.x + S[1]*si.x + S[2]*nj.x + S[3]*sj.x;
	r.y = S[0]*ni.y + S[1]*si.y + S[2]*nj.y + S[3]*sj.y;
	r.z = S[0]*ni.z + S[1]*si.z + S[2]*nj.z + S[3]*sj.z;

	return r;
}
//--------------------------------------------------------------------------------------------------------------------------------
__device__ __host__ inline real3 Calc_ANCF_Point_Slope(
		real3 * ANCF_NodesD,
		real3 * ANCF_SlopesD,
		int indexOfClosestNode,
		real_ s,
		real_ l){
	real_ Sx[4];
	shape_fun_d(Sx, s, l);


	real3 rx;
	real3 ni = ANCF_NodesD[indexOfClosestNode];
	real3 si = ANCF_SlopesD[indexOfClosestNode];
	real3 nj = ANCF_NodesD[indexOfClosestNode + 1];
	real3 sj = ANCF_SlopesD[indexOfClosestNode + 1];

	rx.x = Sx[0]*ni.x + Sx[1]*si.x + Sx[2]*nj.x + Sx[3]*sj.x;
	rx.y = Sx[0]*ni.y + Sx[1]*si.y + Sx[2]*nj.y + Sx[3]*sj.y;
	rx.z = Sx[0]*ni.z + Sx[1]*si.z + Sx[2]*nj.z + Sx[3]*sj.z;

	return rx;
}
//--------------------------------------------------------------------------------------------------------------------------------
__device__ __host__ inline real3 Calc_ANCF_Point_Vel(
		real3 * ANCF_NodesVelD,
		real3 * ANCF_SlopesVelD,
		int indexOfClosestNode,
		real_ s,
		real_ l) {
	real_ S[4];
	shape_fun(S, s, l);


	real3 rt;
	real3 nti = ANCF_NodesVelD[indexOfClosestNode];
	real3 sti = ANCF_SlopesVelD[indexOfClosestNode];
	real3 ntj = ANCF_NodesVelD[indexOfClosestNode + 1];
	real3 stj = ANCF_SlopesVelD[indexOfClosestNode + 1];

	rt.x = S[0]*nti.x + S[1]*sti.x + S[2]*ntj.x + S[3]*stj.x;
	rt.y = S[0]*nti.y + S[1]*sti.y + S[2]*ntj.y + S[3]*stj.y;
	rt.z = S[0]*nti.z + S[1]*sti.z + S[2]*ntj.z + S[3]*stj.z;

	return rt;
}
//--------------------------------------------------------------------------------------------------------------------------------
__device__ __host__ inline real3 Calc_ANCF_Point_Omega(
		real3 * ANCF_NodesVelD,
		real3 * ANCF_SlopesVelD,
		int indexOfClosestNode,
		real_ s,
		real_ l){
	real_ Sx[4];
	shape_fun_d(Sx, s, l);


	real3 omega;
	real3 nti = ANCF_NodesVelD[indexOfClosestNode];
	real3 sti = ANCF_SlopesVelD[indexOfClosestNode];
	real3 ntj = ANCF_NodesVelD[indexOfClosestNode + 1];
	real3 stj = ANCF_SlopesVelD[indexOfClosestNode + 1];

	omega.x = Sx[0]*nti.x + Sx[1]*sti.x + Sx[2]*ntj.x + Sx[3]*stj.x;
	omega.y = Sx[0]*nti.y + Sx[1]*sti.y + Sx[2]*ntj.y + Sx[3]*stj.y;
	omega.z = Sx[0]*nti.z + Sx[1]*sti.z + Sx[2]*ntj.z + Sx[3]*stj.z;

	return omega;
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the fluid particles' properties, i.e. velocity, density, pressure, position
__global__ void UpdateKernelFluid(real3 * posRadD, real4 * velMasD, real3 * vel_XSPH_D, real4 * rhoPresMuD, real4 * derivVelRhoD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	index += updatePortionD.x; // updatePortionD = [start, end] index of the update portion
	if (index >= updatePortionD.y) {
		return;
	}
	real3 vel_XSPH = vel_XSPH_D[index];
	// 1*** let's tweak a little bit :)
	if (length(vel_XSPH) > .2 * HSML / dTD) {
		vel_XSPH *= ( .2 * HSML / dTD ) / length(vel_XSPH);
	}
	// 1*** end tweak
	real3 posRad = posRadD[index];
	real3 updatedPositon = posRad + vel_XSPH * dTD;
	posRadD[index] = updatedPositon; //posRadD updated

	real4 derivVelRho = derivVelRhoD[index];
	real4 velMas = velMasD[index];
	real3 updatedVelocity = R3(velMas + derivVelRho * dTD);
	// 2*** let's tweak a little bit :)
	if (length(updatedVelocity) > .2 * HSML / dTD) {
		updatedVelocity *= ( .2 * HSML / dTD ) / length(updatedVelocity);
	}
	// 2*** end tweak
	velMasD[index] = R4(updatedVelocity, /*rho2 / rhoPresMu.x * */velMas.w); //velMasD updated

	real4 rhoPresMu = rhoPresMuD[index];
	real_ rho2 = rhoPresMu.x + derivVelRho.w * dTD; //rho update. (i.e. rhoPresMu.x), still not wriiten to global matrix
	rhoPresMu.y = Eos(rho2, rhoPresMu.w);
	rhoPresMu.x = rho2;
	rhoPresMuD[index] = rhoPresMu; //rhoPresMuD updated
}
//--------------------------------------------------------------------------------------------------------------------------------
//copies the sortedVelXSPH to velXSPH according to indexing
__global__ void Copy_SortedVelXSPH_To_VelXSPH(real3 * vel_XSPH_D, real3 * vel_XSPH_Sorted_D, uint * m_dGridMarkerIndex, int numMarkers) {
	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numMarkers) return;
	vel_XSPH_D[m_dGridMarkerIndex[index]] = vel_XSPH_Sorted_D[index];
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the fluid particles' properties, i.e. velocity, density, pressure, position
__global__ void UpdateKernelBoundary(real3 * posRadD, real4 * velMasD, real4 * rhoPresMuD, real4 * derivVelRhoD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	index += updatePortionD.x; // updatePortionD = [start, end] index of the update portion
	if (index >= updatePortionD.y) {
		return;
	}

	real4 derivVelRho = derivVelRhoD[index];
	real4 rhoPresMu = rhoPresMuD[index];
	real_ rho2 = rhoPresMu.x + derivVelRho.w * dTD; //rho update. (i.e. rhoPresMu.x), still not wriiten to global matrix
	rhoPresMu.y = Eos(rho2, rhoPresMu.w);
	rhoPresMu.x = rho2;
	rhoPresMuD[index] = rhoPresMu; //rhoPresMuD updated
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along x
__global__ void ApplyPeriodicBoundaryXKernel(real3 * posRadD, real4 * rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numAllMarkersD) {
		return;
	}
	real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	} //no need to do anything if it is a boundary particle
	real3 posRad = posRadD[index];
	if (posRad.x > cMaxD.x) {
		posRad.x -= (cMaxD.x - cMinD.x);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y + bodyForce4.x * (cMaxD.x - cMinD.x);
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
	if (posRad.x < cMinD.x) {
		posRad.x += (cMaxD.x - cMinD.x);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y - bodyForce4.x * (cMaxD.x - cMinD.x);
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along y
__global__ void ApplyPeriodicBoundaryYKernel(real3 * posRadD, real4 * rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numAllMarkersD) {
		return;
	}
	real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	} //no need to do anything if it is a boundary particle
	real3 posRad = posRadD[index];
	if (posRad.y > cMaxD.y) {
		posRad.y -= (cMaxD.y - cMinD.y);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y + bodyForce4.y * (cMaxD.y - cMinD.y);
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
	if (posRad.y < cMinD.y) {
		posRad.y += (cMaxD.y - cMinD.y);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y - bodyForce4.y * (cMaxD.y - cMinD.y);
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
}

//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along z
__global__ void ApplyPeriodicBoundaryZKernel(real3 * posRadD, real4 * rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numAllMarkersD) {
		return;
	}
	real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	} //no need to do anything if it is a boundary particle
	real3 posRad = posRadD[index];
	if (posRad.z > cMaxD.z) {
		posRad.z -= (cMaxD.z - cMinD.z);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y + bodyForce4.z * (cMaxD.z - cMinD.z);
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
	if (posRad.z < cMinD.z) {
		posRad.z += (cMaxD.z - cMinD.z);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y - bodyForce4.z * (cMaxD.z - cMinD.z);
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along x, for ridid bodies
__global__ void ApplyPeriodicBoundaryXKernel_RigidBodies(real3 * posRigidD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numRigidBodiesD) {
		return;
	}
	real3 posRigid = posRigidD[index];
	if (posRigid.x > cMaxD.x) {
		posRigid.x -= (cMaxD.x - cMinD.x);
		posRigidD[index] = posRigid;
		return;
	}
	if (posRigid.x < cMinD.x) {
		posRigid.x += (cMaxD.x - cMinD.x);
		posRigidD[index] = posRigid;
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along x, for ridid bodies
__global__ void ApplyPeriodicBoundaryYKernel_RigidBodies(real3 * posRigidD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numRigidBodiesD) {
		return;
	}
	real3 posRigid = posRigidD[index];
	if (posRigid.y > cMaxD.y) {
		posRigid.y -= (cMaxD.y - cMinD.y);
		posRigidD[index] = posRigid;
		return;
	}
	if (posRigid.y < cMinD.y) {
		posRigid.y += (cMaxD.y - cMinD.y);
		posRigidD[index] = posRigid;
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along x, for ridid bodies
__global__ void ApplyPeriodicBoundaryZKernel_RigidBodies(real3 * posRigidD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numRigidBodiesD) {
		return;
	}
	real3 posRigid = posRigidD[index];
	if (posRigid.z > cMaxD.z) {
		posRigid.z -= (cMaxD.z - cMinD.z);
		posRigidD[index] = posRigid;
		return;
	}
	if (posRigid.z < cMinD.z) {
		posRigid.z += (cMaxD.z - cMinD.z);
		posRigidD[index] = posRigid;
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//related to post processing of Segre-Silberberg. Distribution thing!
__global__ void PassesFromTheEnd_Kernel(
		real3 * posRigidD,
		uint * radialPositions,
		uint * radialPosCounter,
		real2 pipeCenter,
		real_ dR) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numRigidBodiesD) {
		return;
	}
	real3 posRigid = posRigidD[index];
	if ( (posRigid.x > cMaxD.x) || (posRigid.x < cMinD.x) ) {													//assuming the fluid flows in the positive x direction
		real_ r = length(R2(posRigid.y, posRigid.z) - pipeCenter);
		uint radPosition = int(r / dR);
		radialPositions[index] = radPosition;
		radialPosCounter[index] = 1;
			//printf("passed. r %f  dR %f    r/dR %f    radial_pos: %d",  r, dR , r/dR, radPosition);
		return;
	}
	//syncthreads();
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void AddToCumulutaiveNumberOfPasses(
		int * distributionD,
		uint * dummy_radialPosition,
		uint * radialPosCounter_Cumulative,
		int numberOfSections) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numberOfSections) {
		return;
	}
	uint radPosition = dummy_radialPosition[index];
	uint distributionCumul = radialPosCounter_Cumulative[index];
	if (radPosition < numberOfSections) {
		//if (distributionCumul > 0) printf("radPositon %d\n", radPosition);
		distributionD[radPosition] += distributionCumul;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void SumSurfaceInteractionForces(real3 * totalForcesRigid3, real4 * totalSurfaceInteractionRigid4, real4 * velMassRigidD) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numRigidBodiesD) {
		return;
	}
	real4 dummyVelMas = velMassRigidD[rigidSphereA];
	real3 derivRigid = solid_SPH_massD / dummyVelMas.w * R3(totalSurfaceInteractionRigid4[rigidSphereA]);
	//** tweak 3
	if (length(derivRigid) > .2 * HSML / (dTD * dTD)) {
			derivRigid *= ( .2 * HSML / (dTD * dTD) ) / length(derivRigid);
	}
	//** end tweak
	totalForcesRigid3[rigidSphereA] = derivRigid;
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void CalcTorqueShare(real3* torqueMarkersD, real4* derivVelRhoD, real3* posRadD, int* rigidIdentifierD, real3* posRigidD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint rigidMarkerIndex = index + startRigidMarkersD;
	if (index >= numRigid_SphMarkersD) {
		return;
	}
	real3 dist3 = Distance(posRadD[rigidMarkerIndex], posRigidD[rigidIdentifierD[index]]);
	torqueMarkersD[index] = cross(dist3, R3(derivVelRhoD[rigidMarkerIndex]));
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void MapForcesOnNodes(
		real3* flexNodesForcesAllMarkers1,
		real3* flexNodesForcesAllMarkers2,
		int* flexIdentifierD,
		int2* ANCF_ReferenceArrayNodesOnBeamsD,
		int* ANCF_NumMarkers_Per_BeamD,
		int* ANCF_NumMarkers_Per_Beam_CumulD, //exclusive scan
//		int* ANCF_NumNodesMultMarkers_Per_BeamD,
		int* ANCF_NumNodesMultMarkers_Per_Beam_CumulD, //exclusive scan
		real_* flexParametricDistD,
		real_* ANCF_Beam_LengthD,
		real4* derivVelRhoD)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numFlex_SphMarkersD) {
		return;
	}
	uint absMarkerIndex = index + startFlexMarkersD;
	real_ s = flexParametricDistD[index];

	real3 derivVel = R3( derivVelRhoD[absMarkerIndex] );
	real3 markerForce = solid_SPH_massD * derivVel;

//	Map Marker Force to ANCF Nodes, gives you as many forces as the number of nodes per beam
//	F0, F1, ..., F(m-1) : Forces on nodes 0, 1, 2, ..., m-1
//	Fi ---> flexNodesForces[numSavedForcesSoFar + (i * numMarkersOnThisBeam + markerIndexOnThisBeam)];
//	...

	int flexBodyIndex = flexIdentifierD[index];
	real_ l = ANCF_Beam_LengthD[flexBodyIndex];


	int numFlexMarkersPreviousBeamsTotal = ANCF_NumMarkers_Per_Beam_CumulD[flexBodyIndex];
	int markerIndexOnThisBeam = index - numFlexMarkersPreviousBeamsTotal;
	int numMarkersOnThisBeam = ANCF_NumMarkers_Per_BeamD[flexBodyIndex];
	int numSavedForcesSoFar = ANCF_NumNodesMultMarkers_Per_Beam_CumulD[flexBodyIndex];

	int2 nodesInterval = ANCF_ReferenceArrayNodesOnBeamsD[flexBodyIndex];
	int nNodes = nodesInterval.y - nodesInterval.x;
	int indexOfClosestNode = int(s / l * nNodes);
	if (indexOfClosestNode == nNodes) indexOfClosestNode--;


	real_ f_a[12] = {0};
	Applied_Force(f_a, s, l, markerForce);
	//left node
	flexNodesForcesAllMarkers1[numSavedForcesSoFar + indexOfClosestNode * numMarkersOnThisBeam + markerIndexOnThisBeam] = R3(f_a[0], f_a[1], f_a[2]);
	flexNodesForcesAllMarkers2[numSavedForcesSoFar + indexOfClosestNode * numMarkersOnThisBeam + markerIndexOnThisBeam] = R3(f_a[3], f_a[4], f_a[5]);
	//right node
	flexNodesForcesAllMarkers1[numSavedForcesSoFar + (indexOfClosestNode + 1) * numMarkersOnThisBeam + markerIndexOnThisBeam] = R3(f_a[6], f_a[7], f_a[8]);
	flexNodesForcesAllMarkers2[numSavedForcesSoFar + (indexOfClosestNode + 1) * numMarkersOnThisBeam + markerIndexOnThisBeam] = R3(f_a[9], f_a[10], f_a[11]);


}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Populate_RigidSPH_MeshPos_LRF_kernel(
		real3* rigidSPH_MeshPos_LRF_D,
		real3* posRadD,
		int* rigidIdentifierD,
		real3* posRigidD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint rigidMarkerIndex = index + startRigidMarkersD; // updatePortionD = [start, end] index of the update portion
	if (index >= numRigid_SphMarkersD) {
		return;
	}
	real3 dist3 = posRadD[rigidMarkerIndex] - posRigidD[rigidIdentifierD[index]];
	rigidSPH_MeshPos_LRF_D[index] = dist3;
}
//--------------------------------------------------------------------------------------------------------------------------------

__global__ void Populate_FlexSPH_MeshPos_LRF_kernel(
		real3* flexSPH_MeshPos_LRF_D,
		real3 * posRadD,
		int* flexIdentifierD,
		real_* flexParametricDistD,
		real_* ANCF_Beam_LengthD,
		int2* ANCF_ReferenceArrayNodesOnBeamsD,
		real3 * ANCF_NodesD,
		real3 * ANCF_SlopesD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numFlex_SphMarkersD) {
		return;
	}
	uint absMarkerIndex = index + startFlexMarkersD; // updatePortionD = [start, end] index of the update portion
	real_ s = flexParametricDistD[index];
	int flexBodyIndex = flexIdentifierD[index];
	real_ l = ANCF_Beam_LengthD[flexBodyIndex];
	int2 nodesInterval = ANCF_ReferenceArrayNodesOnBeamsD[flexBodyIndex];
	int nNodes = nodesInterval.y - nodesInterval.x;

	int indexOfClosestNode = int(s / l * nNodes);
	if (indexOfClosestNode == nNodes) indexOfClosestNode--;

	real3 beamPointPos = Calc_ANCF_Point_Pos(ANCF_NodesD, ANCF_SlopesD, indexOfClosestNode, s, l); //interpolation using ANCF beam, cubic hermit equation

	real3 dist3 = posRadD[absMarkerIndex] - beamPointPos;
	flexSPH_MeshPos_LRF_D[index] = dist3;
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Populate_FlexSPH_MeshSlope_LRF_kernel(
		real3* flexSPH_MeshSlope_Initial_D,
		int* flexIdentifierD,
		real_* flexParametricDistD,
		real_* ANCF_Beam_LengthD,
		int2* ANCF_ReferenceArrayNodesOnBeamsD,
		real3 * ANCF_NodesD,
		real3 * ANCF_SlopesD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numFlex_SphMarkersD) {
		return;
	}
	uint absMarkerIndex = index + startFlexMarkersD; // updatePortionD = [start, end] index of the update portion
	real_ s = flexParametricDistD[index];
	int flexBodyIndex = flexIdentifierD[index];
	real_ l = ANCF_Beam_LengthD[flexBodyIndex];
	int2 nodesInterval = ANCF_ReferenceArrayNodesOnBeamsD[flexBodyIndex];
	int nNodes = nodesInterval.y - nodesInterval.x;

	int indexOfClosestNode = int(s / l * nNodes);
	if (indexOfClosestNode == nNodes) indexOfClosestNode--;

	real3 beamPointSlope = Calc_ANCF_Point_Slope(ANCF_NodesD, ANCF_SlopesD, indexOfClosestNode, s, l); //interpolation using ANCF beam, cubic hermit equation
	flexSPH_MeshSlope_Initial_D[index] = beamPointSlope;
}

//--------------------------------------------------------------------------------------------------------------------------------
//the rigid body torque has been calculated in global RF. This kernel maps it to local RF to be appropriate for the formulas
//local torque = T' = A' * T
__global__ void MapTorqueToLRFKernel(real3 * AD1, real3 * AD2, real3 * AD3, real3 * totalTorque3, real3 * LF_totalTorque3) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numRigidBodiesD) {
		return;
	}
	real3 totalTorqueGRF = totalTorque3[rigidSphereA];
	LF_totalTorque3[rigidSphereA] = AD1[rigidSphereA] * totalTorqueGRF.x + AD2[rigidSphereA] * totalTorqueGRF.y
			+ AD3[rigidSphereA] * totalTorqueGRF.z;
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the rigid body particles
__global__ void UpdateKernelRigidTranstalation(real3 * totalForcesRigid3, real3 * posRigidD, real3 * posRigidCumulativeD, real4 * velMassRigidD) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numRigidBodiesD) {
		return;
	}

	real3 dummyPos = posRigidD[rigidSphereA];
	real4 dummyVelMas = velMassRigidD[rigidSphereA];

	real3 derivV_SPH = totalForcesRigid3[rigidSphereA]; //in fact, totalBodyForce4 is originially sum of dV/dt of sph particles and should be multiplied by m to produce force. gravity is applied in the force kernel

	real3 deltaPos = R3(dummyVelMas) * dTD;
	dummyPos += deltaPos;
	posRigidD[rigidSphereA] = dummyPos;
	posRigidCumulativeD[rigidSphereA] += deltaPos;

	real3 deltaVel = derivV_SPH * dTD;
	dummyVelMas += R4(deltaVel, 0);
	velMassRigidD[rigidSphereA] = dummyVelMas;
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the rigid body particles
__global__ void UpdateKernelRigidTranstalationBeta(real3 * totalForcesRigid3, real3 * posRigidD, real3 * posRigidCumulativeD, real4 * velMassRigidD) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numRigidBodiesD) {
		return;
	}

	real3 dummyPos = posRigidD[rigidSphereA];
	real4 dummyVelMas = velMassRigidD[rigidSphereA];

	real3 derivV_SPH = totalForcesRigid3[rigidSphereA]; //in fact, totalBodyForce4 is originially sum of dV/dt of sph particles and should be multiplied by m to produce force. gravity is applied in the force kernel
	derivV_SPH.y = 0;
	derivV_SPH.z = 0;

	real3 deltaPos = R3(dummyVelMas) * dTD;
	dummyPos += deltaPos;
	posRigidD[rigidSphereA] = dummyPos;
	posRigidCumulativeD[rigidSphereA] += deltaPos;

	real3 deltaVel = derivV_SPH * dTD;
	dummyVelMas += R4(deltaVel, 0);
	velMassRigidD[rigidSphereA] = dummyVelMas;
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the rigid body Quaternion of Rotation
// A is rotation matrix, A = [AD1; AD2; AD3]
__global__ void UpdateRigidBodyQuaternion_kernel(real4 * qD, real3 * omegaLRF_D) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numRigidBodiesD) {
		return;
	}
	real3 omega = omegaLRF_D[rigidSphereA];
	real4 q = qD[rigidSphereA];
	real4 qDot = 0.5 * (
			omega.x * R4(-(q.y), q.x, q.w, -(q.z)) + omega.y * R4(-(q.z), -(q.w), q.x, q.y) + omega.z * R4(-(q.w), q.z, -(q.y), q.x)
	);

	q += dTD * qDot;
	q *= (1.0f / length(q));
	qD[rigidSphereA] = q;
}
//--------------------------------------------------------------------------------------------------------------------------------
// first comp of q is rotation, last 3 components are axis of rot
__device__ inline void RotationMatirixFromQuaternion_kernelD(real3 & AD1, real3 & AD2, real3 & AD3, const real4 & q) {
	AD1 = 2 * R3(0.5f - q.z * q.z - q.w * q.w, q.y * q.z - q.x * q.w, q.y * q.w + q.x * q.z);
	AD2 = 2 * R3(q.y * q.z + q.x * q.w, 0.5f - q.y * q.y - q.w * q.w, q.z * q.w - q.x * q.y);
	AD3 = 2 * R3(q.y * q.w - q.x * q.z, q.z * q.w + q.x * q.y, 0.5f - q.y * q.y - q.z * q.z);
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the rigid body Rotation
// A is rotation matrix, A = [AD1; AD2; AD3], first comp of q is rotation, last 3 components are axis of rot
// in wikipedia, last quat comp is the angle, in my version, first one is the angle.
// here is the mapping between wikipedia (g) and mine (q): [gx, gy, gz, gw] = [qy, qz, qw, qx]
__global__ void RotationMatirixFromQuaternion_kernel(real3 * AD1, real3 * AD2, real3 * AD3, real4 * qD) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numRigidBodiesD) {
		return;
	}
	real4 q = qD[rigidSphereA];
	AD1[rigidSphereA] = 2 * R3(0.5f - q.z * q.z - q.w * q.w, q.y * q.z - q.x * q.w, q.y * q.w + q.x * q.z);
	AD2[rigidSphereA] = 2 * R3(q.y * q.z + q.x * q.w, 0.5f - q.y * q.y - q.w * q.w, q.z * q.w - q.x * q.y);
	AD3[rigidSphereA] = 2 * R3(q.y * q.w - q.x * q.z, q.z * q.w + q.x * q.y, 0.5f - q.y * q.y - q.z * q.z);
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void UpdateRigidBodyAngularVelocity_kernel(
		real3 * LF_totalTorque3,
		real3 * jD1,
		real3 * jD2,
		real3 * jInvD1,
		real3 * jInvD2,
		real3 * omegaLRF_D) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numRigidBodiesD) {
		return;
	}

	real3 omega3 = omegaLRF_D[rigidSphereA];
	real3 j1 = jD1[rigidSphereA];
	real3 j2 = jD2[rigidSphereA];
	//printf("j j %f %f %f %f %f %f\n", j1.x, j1.y, j1.z, j2.x, j2.y, j2.z);
	real3 torquingTerm;
	torquingTerm.x = (-omega3.z * j1.y + omega3.y * j1.z) * omega3.x + (-omega3.z * j2.x + omega3.y * j2.y) * omega3.y
			+ (-omega3.z * j2.y + omega3.y * j2.z) * omega3.z;
	torquingTerm.y = (omega3.z * j1.x - omega3.x * j1.z) * omega3.x + (omega3.z * j1.y - omega3.x * j2.y) * omega3.y
			+ (omega3.z * j1.z - omega3.x * j2.z) * omega3.z;
	torquingTerm.z = (-omega3.y * j1.x + omega3.x * j1.y) * omega3.x + (-omega3.y * j1.y + omega3.x * j2.x) * omega3.y
			+ (-omega3.y * j1.z + omega3.x * j2.y) * omega3.z;

	torquingTerm = solid_SPH_massD * LF_totalTorque3[rigidSphereA] - torquingTerm;
	//*** from this point j1 and j2 will represent the j_Inverse
	j1 = jInvD1[rigidSphereA];
	j2 = jInvD2[rigidSphereA];
	//printf("j j %f %f %f %f %f %f\n", j1.x, j1.y, j1.z, j2.x, j2.y, j2.z);
	real3 omegaDot3 = torquingTerm.x * j1 + torquingTerm.y * R3(j1.y, j2.x, j2.y) + torquingTerm.z * R3(j1.z, j2.y, j2.z);
//	//	*** for 2D motion
//		omegaDot3.x = 0;
//		omegaDot3.z = 0;

	omega3 += omegaDot3 * dTD;
	omegaLRF_D[rigidSphereA] = omega3;
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the rigid body particles
__global__ void UpdateRigidMarkersPosition(
		real3 * posRadD,
		real4 * velMasD,
		const real3 * rigidSPH_MeshPos_LRF_D,
		const int * rigidIdentifierD,
		real3 * posRigidD,
		real4 * velMassRigidD,
		real3 * omegaLRF_D,
		real3 * AD1,
		real3 * AD2,
		real3 * AD3) {

	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numRigid_SphMarkersD) {
		return;
	}
	uint rigidMarkerIndex = index + startRigidMarkersD; // updatePortionD = [start, end] index of the update portion
	int rigidBodyIndex = rigidIdentifierD[index];

	real3 a1, a2, a3;
	a1 = AD1[rigidBodyIndex];
	a2 = AD2[rigidBodyIndex];
	a3 = AD3[rigidBodyIndex];

	real3 rigidSPH_MeshPos_LRF = rigidSPH_MeshPos_LRF_D[index];

	//position
	real3 p_Rigid = posRigidD[rigidBodyIndex];
	posRadD[rigidMarkerIndex] = p_Rigid + R3(dot(a1, rigidSPH_MeshPos_LRF), dot(a2, rigidSPH_MeshPos_LRF), dot(a3, rigidSPH_MeshPos_LRF));

	//velociy
	real4 vM = velMasD[rigidMarkerIndex];
	real4 vM_Rigid = velMassRigidD[rigidBodyIndex];
	real3 omega3 = omegaLRF_D[rigidBodyIndex];
	real3 omegaCrossS = cross(omega3, rigidSPH_MeshPos_LRF);
	velMasD[rigidMarkerIndex] = R4(R3(vM_Rigid) + R3(dot(a1, omegaCrossS), dot(a2, omegaCrossS), dot(a3, omegaCrossS)), vM.w);
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the flex body markers
__global__ void UpdateFlexMarkersPosition(
		real3 * posRadD,
		real4 * velMasD,
		int* flexIdentifierD,
		real3* flexSPH_MeshPos_LRF_D,
		real3* flexSPH_MeshSlope_Initial_D,
		real_* flexParametricDistD,
		real_* ANCF_Beam_LengthD,
		int2* ANCF_ReferenceArrayNodesOnBeamsD,
		real3 * ANCF_NodesD,
		real3 * ANCF_SlopesD,
		real3 * ANCF_NodesVelD,
		real3 * ANCF_SlopesVelD) {

	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numFlex_SphMarkersD) {
		return;
	}
	uint absMarkerIndex = index + startFlexMarkersD; // updatePortionD = [start, end] index of the update portion
	real_ s = flexParametricDistD[index];
	int flexBodyIndex = flexIdentifierD[index];
	real_ l = ANCF_Beam_LengthD[flexBodyIndex];
	int2 nodesInterval = ANCF_ReferenceArrayNodesOnBeamsD[flexBodyIndex];
	int nNodes = nodesInterval.y - nodesInterval.x;

	int indexOfClosestNode = int(s / l * nNodes);
	if (indexOfClosestNode == nNodes) indexOfClosestNode--;

	real3 beamPointPos = Calc_ANCF_Point_Pos(ANCF_NodesD, ANCF_SlopesD, indexOfClosestNode, s, l); //interpolation using ANCF beam, cubic hermit equation
	real3 beamPointSlope = Calc_ANCF_Point_Slope(ANCF_NodesD, ANCF_SlopesD, indexOfClosestNode, s, l); //interpolation using ANCF beam, cubic hermit equation
	real3 beamPointOmega;

	real3 beamPointVel = Calc_ANCF_Point_Vel(ANCF_NodesD, ANCF_SlopesD, ANCF_NodesVelD, ANCF_SlopesVelD, indexOfClosestNode, s, l); //interpolation using ANCF beam, cubic hermit equation

	real3 dist3 = flexSPH_MeshPos_LRF_D[index];
	real3 beamPointSlopeInitial = flexSPH_MeshSlope_Initial_D[index];
	//Important Important Important Important Important Important Important Important Important
	//Important Important Important Important Important Important Important Important Important
	//Important Important Important Important Important Important Important Important Important
	// Assumed Calc_ANCF_Point_Slope returns the unit vector. theta calculation is based on this assumption. Also cross product
	real_ theta = acos(dot(beamPointSlopeInitial, beamPointSlope));
	real3 n3 = cross(beamPointSlopeInitial, beamPointSlope);
	n3 /= length(n3);
	real4 q = R4(cos(0.5 * theta),
			n3.x * sin(0.5 * theta), n3.y * sin(0.5 * theta), n3.z * sin(0.5 * theta));
	real3 A1, A2, A3;
	RotationMatirixFromQuaternion_kernelD(A1, A2, A3, q);
	posRadD[absMarkerIndex] = beamPointPos + R3(dot(A1, dist3), dot(A2, dist3), dot(A3, dist3));

	//ask Radu
	real3 absOmega = Calc_ANCF_Point_Omega(ANCF_NodesD, ANCF_SlopesD, ANCF_NodesVelD, ANCF_SlopesVelD, indexOfClosestNode, s, l); //interpolation using ANCF beam, cubic hermit equation
	velMasD[absMarkerIndex] = beamPointVel + cross(absOmega, dist3);
}
////--------------------------------------------------------------------------------------------------------------------------------
void MakeRigidIdentifier(
		thrust::device_vector<int> & rigidIdentifierD,
		int numRigidBodies, int startRigidMarkers, const thrust::host_vector<int3> & referenceArray)
{
	if (numRigidBodies > 0) {
		for (int rigidSphereA = 0; rigidSphereA < numRigidBodies; rigidSphereA++) {
			int3 referencePart = referenceArray[2 + rigidSphereA];
			if (referencePart.z != 1) {
				printf("error in accessing rigid bodies. Reference array indexing is wrong\n");
				return;
			}
			int2 updatePortion = I2(referencePart); //first two component of the referenceArray denote to the fluid and boundary particles
			thrust::fill(rigidIdentifierD.begin() + (updatePortion.x - startRigidMarkers),
					rigidIdentifierD.begin() + (updatePortion.y - startRigidMarkers), rigidSphereA);
		}
	}
}
////--------------------------------------------------------------------------------------------------------------------------------

////; flexIdentifier is not of the size of total flex bodies. Here, apparently, it is of the size of total markers!!!!!!!!!!!!!!!!!!!!!!!!
void MakeFlexIdentifier(
		thrust::device_vector<int> & flexIdentifierD,
		int numFlexBodies, int numFlBcRigid, int startFlexMarkers, const thrust::host_vector<int3> & referenceArray)
{
	if (numFlexBodies > 0) {
		for (int flexIdx = 0; flexIdx < numFlexBodies; flexIdx++) {
			int3 referencePart = referenceArray[numFlBcRigid + flexIdx];
			if (referencePart.z != 1) {
				printf("error in accessing rigid bodies. Reference array indexing is wrong\n");
				return;
			}
			int2 updatePortion = I2(referencePart); //first two component of the referenceArray denote to the fluid and boundary particles
			thrust::fill(flexIdentifierD.begin() + (updatePortion.x - startFlexMarkers),
					flexIdentifierD.begin() + (updatePortion.y - startFlexMarkers), flexIdx);
		}
	}
}
////--------------------------------------------------------------------------------------------------------------------------------
void Calc_NumNodesMultMarkers_Per_Beam(
		thrust::device_vector<int> & ANCF_NumNodesMultMarkers_Per_BeamD,
		const thrust::device_vector<int> & ANCF_NumMarkers_Per_BeamD,
		const thrust::host_vector<int2> & ANCF_ReferenceArrayNodesOnBeams,
		int numFlexBodies)
{
	if (numFlexBodies > 0) {
		for (int flexIdx = 0; flexIdx < numFlexBodies; flexIdx++) {
			int2 flexPortion = ANCF_ReferenceArrayNodesOnBeams[flexIdx];
			int numNodes = flexPortion.y - flexPortion.x;
			ANCF_NumNodesMultMarkers_Per_BeamD[flexIdx] = numNodes * ANCF_NumMarkers_Per_BeamD[flexIdx];
		}
	}
}
////--------------------------------------------------------------------------------------------------------------------------------
void Calc_mapEachMarkerOnAllBeamNodes_IdentifierD(
		thrust::device_vector<int2> & flexMapEachMarkerOnAllBeamNodesD,
		const thrust::device_vector<int> & ANCF_NumNodesMultMarkers_Per_Beam_CumulD,
		const thrust::device_vector<int> & ANCF_NumMarkers_Per_BeamD,
		const thrust::host_vector<int2> & ANCF_ReferenceArrayNodesOnBeams,
		int numFlexBodies)
{
	if (numFlexBodies > 0) {
		for (int flexIdx = 0; flexIdx < numFlexBodies; flexIdx++) {
			int numMarkersOnThisBeam = ANCF_NumMarkers_Per_BeamD[flexIdx];
			int2 flexPortion = ANCF_ReferenceArrayNodesOnBeams[flexIdx];
			int numNodes = flexPortion.y - flexPortion.x;

			int2 startWrite2 = ANCF_NumNodesMultMarkers_Per_Beam_CumulD[flexIdx];
			for (int i = 0; i < numNodes; i++) {
				int2 flexIdx_nodeIdx_pair = I2(flexIdx, i);
				int2 writeInterval = I2(startWrite2.x + i * numMarkersOnThisBeam, startWrite2.x + (i + 1) * numMarkersOnThisBeam);
				thrust::fill(flexMapEachMarkerOnAllBeamNodesD.begin() + writeInterval.x,
						flexMapEachMarkerOnAllBeamNodesD.begin() + writeInterval.y, flexIdx_nodeIdx_pair);
			}
		}
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
Update_ANCF_Beam(
		thrust::device_vector<real3> & ANCF_NodesD,
		thrust::device_vector<real3> & ANCF_SlopesD,
		thrust::device_vector<real3> & ANCF_NodesVelD,
		thrust::device_vector<real3> & ANCF_SlopesVelD,
		const thrust::device_vector<real3> & flex_FSI_NodesForces1,
		const thrust::device_vector<real3> & flex_FSI_NodesForces2,
		const thrust::device_vector<real_> & ANCF_Beam_LengthD);
//--------------------------------------------------------------------------------------------------------------------------------
void MapSPH_ToGrid(
		real_ resolution,
		int3 & cartesianGridDims,
		thrust::host_vector<real4> & rho_Pres_CartH,
		thrust::host_vector<real4> & vel_VelMag_CartH,
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real4> & rhoPresMuD,
		int numAllMarkers,
		SimParams paramsH) {
//	real3* m_dSortedPosRad;
//	real4* m_dSortedVelMas;
//	real4* m_dSortedRhoPreMu;
//	uint* m_dCellStart; // index of start of each cell in sorted list
//	uint* m_dCellEnd; // index of end of cell

	int3 SIDE = paramsH.gridSize;
	uint m_numGridCells = SIDE.x * SIDE.y * SIDE.z; //m_gridSize = SIDE
	//TODO here

	// calculate grid hash
	thrust::device_vector<real3> m_dSortedPosRad(numAllMarkers);
	thrust::device_vector<real4> m_dSortedVelMas(numAllMarkers);
	thrust::device_vector<real4> m_dSortedRhoPreMu(numAllMarkers);

	thrust::device_vector<uint> m_dGridMarkerHash(numAllMarkers);
	thrust::device_vector<uint> m_dGridMarkerIndex(numAllMarkers);

	thrust::device_vector<uint> m_dCellStart(m_numGridCells);
	thrust::device_vector<uint> m_dCellEnd(m_numGridCells);

	// calculate grid hash
	calcHash(U1CAST(m_dGridMarkerHash), U1CAST(m_dGridMarkerIndex), R3CAST(posRadD), numAllMarkers);

	thrust::sort_by_key(m_dGridMarkerHash.begin(), m_dGridMarkerHash.end(), m_dGridMarkerIndex.begin());

	// reorder particle arrays into sorted order and find start and end of each cell
	reorderDataAndFindCellStart(U1CAST(m_dCellStart), U1CAST(m_dCellEnd), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu), U1CAST(m_dGridMarkerHash),
			U1CAST(m_dGridMarkerIndex), TCAST(posRadD), R4CAST(velMasD), R4CAST(rhoPresMuD), numAllMarkers, m_numGridCells);

	//real_ resolution = 8 * paramsH.markerRadius;
	cartesianGridDims = I3(paramsH.boxDims / resolution) + I3(1);
//	printf("^^^ bodDim %f %f %f, GridDim %d %d %d, resolution %f \n", paramsH.boxDims.x, paramsH.boxDims.y, paramsH.boxDims.z, cartesianGridDims.x,
//			cartesianGridDims.y, cartesianGridDims.z, resolution);
	uint cartesianGridSize = cartesianGridDims.x * cartesianGridDims.y * cartesianGridDims.z;
	thrust::device_vector<real4> rho_Pres_CartD(cartesianGridSize);
	thrust::device_vector<real4> vel_VelMag_CartD(cartesianGridSize);

	CalcCartesianData(R4CAST(rho_Pres_CartD), R4CAST(vel_VelMag_CartD), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu),
			U1CAST(m_dGridMarkerIndex), U1CAST(m_dCellStart), U1CAST(m_dCellEnd), cartesianGridSize, cartesianGridDims, resolution);

//	freeArray(m_dSortedPosRad);
//	freeArray(m_dSortedVelMas);
//	freeArray(m_dSortedRhoPreMu);
	m_dSortedPosRad.clear();
	m_dSortedVelMas.clear();
	m_dSortedRhoPreMu.clear();

	m_dGridMarkerHash.clear();
	m_dGridMarkerIndex.clear();

//	freeArray(m_dCellStart);
//	freeArray(m_dCellEnd);
	m_dCellStart.clear();
	m_dCellEnd.clear();

	rho_Pres_CartH.resize(cartesianGridSize);
	vel_VelMag_CartH.resize(cartesianGridSize);
	thrust::copy(rho_Pres_CartD.begin(), rho_Pres_CartD.end(), rho_Pres_CartH.begin());
	thrust::copy(vel_VelMag_CartD.begin(), vel_VelMag_CartD.end(), vel_VelMag_CartH.begin());

	rho_Pres_CartD.clear();
	vel_VelMag_CartD.clear();
}

//*******************************************************************************************************************************
//builds the neighbors' list of each particle and finds the force on each particle
//calculates the interaction force between 1- fluid-fluid, 2- fluid-solid, 3- solid-fluid particles
//calculates forces from other SPH or solid particles, as wall as boundaries
void ForceSPH(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real3> & vel_XSPH_D,
		thrust::device_vector<real4> & rhoPresMuD,
		thrust::device_vector<uint> & bodyIndexD,
		thrust::device_vector<real4> & derivVelRhoD,
		const thrust::host_vector<int3> & referenceArray,
		int numAllMarkers,
		int3 SIDE,
		real_ dT) {
	// Part1: contact detection #########################################################################################################################
	// grid data for sorting method
//	real3* m_dSortedPosRad;
//	real4* m_dSortedVelMas;
//	real4* m_dSortedRhoPreMu;
//	uint* m_dCellStart; // index of start of each cell in sorted list
//	uint* m_dCellEnd; // index of end of cell

	uint m_numGridCells = SIDE.x * SIDE.y * SIDE.z; //m_gridSize = SIDE
	//TODO here

	// calculate grid hash
	thrust::device_vector<real3> m_dSortedPosRad(numAllMarkers);
	thrust::device_vector<real4> m_dSortedVelMas(numAllMarkers);
	thrust::device_vector<real4> m_dSortedRhoPreMu(numAllMarkers);
	thrust::device_vector<real3> vel_XSPH_Sorted_D(numAllMarkers);

	thrust::device_vector<uint> m_dGridMarkerHash(numAllMarkers);
	thrust::device_vector<uint> m_dGridMarkerIndex(numAllMarkers);

	thrust::device_vector<uint> m_dCellStart(m_numGridCells);
	thrust::device_vector<uint> m_dCellEnd(m_numGridCells);
	// calculate grid hash
	calcHash(U1CAST(m_dGridMarkerHash), U1CAST(m_dGridMarkerIndex), R3CAST(posRadD), numAllMarkers);

	thrust::sort_by_key(m_dGridMarkerHash.begin(), m_dGridMarkerHash.end(), m_dGridMarkerIndex.begin());

	// reorder particle arrays into sorted order and find start and end of each cell
	reorderDataAndFindCellStart(U1CAST(m_dCellStart), U1CAST(m_dCellEnd), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu), U1CAST(m_dGridMarkerHash),
			U1CAST(m_dGridMarkerIndex), TCAST(posRadD), R4CAST(velMasD), R4CAST(rhoPresMuD), numAllMarkers, m_numGridCells);

	//process collisions
	real4 totalFluidBodyForce4 = bodyForce4 + R4(Gravity);
	thrust::fill(derivVelRhoD.begin(), derivVelRhoD.end(), R4(0)); //initialize derivVelRhoD with zero. necessary
	thrust::fill(derivVelRhoD.begin() + referenceArray[0].x, derivVelRhoD.begin() + referenceArray[0].y, totalFluidBodyForce4); //add body force to fluid particles.

	RecalcVelocity_XSPH(R3CAST(vel_XSPH_Sorted_D), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu), U1CAST(m_dGridMarkerIndex), U1CAST(m_dCellStart),
			U1CAST(m_dCellEnd), numAllMarkers, m_numGridCells);

	collide(R4CAST(derivVelRhoD), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R3CAST(vel_XSPH_Sorted_D), R4CAST(m_dSortedRhoPreMu), U1CAST(m_dGridMarkerIndex), U1CAST(m_dCellStart),
			U1CAST(m_dCellEnd), numAllMarkers, m_numGridCells, dT);


	uint nBlock_NumSpheres, nThreads_SphMarkers;
	computeGridSize(numAllMarkers, 256, nBlock_NumSpheres, nThreads_SphMarkers);
	Copy_SortedVelXSPH_To_VelXSPH<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(R3CAST(vel_XSPH_D), R3CAST(vel_XSPH_Sorted_D), U1CAST(m_dGridMarkerIndex), numAllMarkers);
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: Copy_SortedVelXSPH_To_VelXSPH");

	////
	m_dSortedPosRad.clear();
	m_dSortedVelMas.clear();
	m_dSortedRhoPreMu.clear();
	vel_XSPH_Sorted_D.clear();

	m_dGridMarkerHash.clear();
	m_dGridMarkerIndex.clear();

	m_dCellStart.clear();
	m_dCellEnd.clear();
}
//--------------------------------------------------------------------------------------------------------------------------------
void DensityReinitialization(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real4> & rhoPresMuD,
		int numAllMarkers,
		int3 SIDE) {
//	real3* m_dSortedPosRad;
//	real4* m_dSortedVelMas;
//	real4* m_dSortedRhoPreMu;
//	uint* m_dCellStart; // index of start of each cell in sorted list
//	uint* m_dCellEnd; // index of end of cell

	uint m_numGridCells = SIDE.x * SIDE.y * SIDE.z; //m_gridSize = SIDE
	//TODO here

	// calculate grid hash
	thrust::device_vector<real3> m_dSortedPosRad(numAllMarkers);
	thrust::device_vector<real4> m_dSortedVelMas(numAllMarkers);
	thrust::device_vector<real4> m_dSortedRhoPreMu(numAllMarkers);

	thrust::device_vector<uint> m_dGridMarkerHash(numAllMarkers);
	thrust::device_vector<uint> m_dGridMarkerIndex(numAllMarkers);

	thrust::device_vector<uint> m_dCellStart(m_numGridCells);
	thrust::device_vector<uint> m_dCellEnd(m_numGridCells);

	// calculate grid hash
	calcHash(U1CAST(m_dGridMarkerHash), U1CAST(m_dGridMarkerIndex), R3CAST(posRadD), numAllMarkers);

	thrust::sort_by_key(m_dGridMarkerHash.begin(), m_dGridMarkerHash.end(), m_dGridMarkerIndex.begin());

	// reorder particle arrays into sorted order and find start and end of each cell
	reorderDataAndFindCellStart(U1CAST(m_dCellStart), U1CAST(m_dCellEnd), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu), U1CAST(m_dGridMarkerHash),
			U1CAST(m_dGridMarkerIndex), TCAST(posRadD), R4CAST(velMasD), R4CAST(rhoPresMuD), numAllMarkers, m_numGridCells);

	ReCalcDensity(R3CAST(posRadD), R4CAST(velMasD), R4CAST(rhoPresMuD), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu),
			U1CAST(m_dGridMarkerIndex), U1CAST(m_dCellStart), U1CAST(m_dCellEnd), numAllMarkers, m_numGridCells);

	m_dSortedPosRad.clear();
	m_dSortedVelMas.clear();
	m_dSortedRhoPreMu.clear();

	m_dGridMarkerHash.clear();
	m_dGridMarkerIndex.clear();

	m_dCellStart.clear();
	m_dCellEnd.clear();
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the fluid particles by calling UpdateKernelFluid 
void UpdateFluid(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real3> & vel_XSPH_D,
		thrust::device_vector<real4> & rhoPresMuD,
		thrust::device_vector<real4> & derivVelRhoD,
		const thrust::host_vector<int3> & referenceArray,
		real_ dT) {
	int3 referencePortion = referenceArray[0];
	if (referencePortion.z != -1) {
		printf("error in UpdateFluid, accessing non fluid\n");
		return;
	}
	int2 updatePortion = I2(referencePortion);
	//int2 updatePortion = I2(referenceArray[0].x, referenceArray[0].y);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dTD), &dT, sizeof(dT));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(updatePortionD), &updatePortion, sizeof(updatePortion));

	uint nBlock_UpdateFluid, nThreads;
	computeGridSize(updatePortion.y - updatePortion.x, 128, nBlock_UpdateFluid, nThreads);
	UpdateKernelFluid<<<nBlock_UpdateFluid, nThreads>>>(R3CAST(posRadD), R4CAST(velMasD), R3CAST(vel_XSPH_D), R4CAST(rhoPresMuD), R4CAST(derivVelRhoD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelFluid");
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the fluid particles by calling UpdateBoundary
void UpdateBoundary(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real4> & rhoPresMuD,
		thrust::device_vector<real4> & derivVelRhoD,
		const thrust::host_vector<int3> & referenceArray,
		real_ dT) {
	int3 referencePortion = referenceArray[1];
	if (referencePortion.z != 0) {
		printf("error in UpdateBoundary, accessing non boundary\n");
		return;
	}
	int2 updatePortion = I2(referencePortion);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dTD), &dT, sizeof(dT));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(updatePortionD), &updatePortion, sizeof(updatePortion));

	uint nBlock_UpdateFluid, nThreads;
	computeGridSize(updatePortion.y - updatePortion.x, 128, nBlock_UpdateFluid, nThreads);
	UpdateKernelBoundary<<<nBlock_UpdateFluid, nThreads>>>(R3CAST(posRadD), R4CAST(velMasD), R4CAST(rhoPresMuD), R4CAST(derivVelRhoD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelBoundary");
}
//--------------------------------------------------------------------------------------------------------------------------------
void ApplyBoundary(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & rhoPresMuD,
		int numAllMarkers,
		thrust::device_vector<real3> & posRigidD,
		thrust::device_vector<real4> & velMassRigidD,
		int numRigidBodies) {
	uint nBlock_NumSpheres, nThreads_SphMarkers;
	computeGridSize(numAllMarkers, 256, nBlock_NumSpheres, nThreads_SphMarkers);
	ApplyPeriodicBoundaryXKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(R3CAST(posRadD), R4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: ApplyPeriodicBoundaryXKernel");
	// these are useful anyway for out of bound particles
	ApplyPeriodicBoundaryYKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(R3CAST(posRadD), R4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: ApplyPeriodicBoundaryXKernel");
	ApplyPeriodicBoundaryZKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(R3CAST(posRadD), R4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: ApplyPeriodicBoundaryXKernel");
//////////////
	uint nBlock_NumRigids, nThreads_RigidBodies;
	computeGridSize(numRigidBodies, 128, nBlock_NumRigids, nThreads_RigidBodies);

	hipMemcpyToSymbolAsync(HIP_SYMBOL(numRigidBodiesD), &numRigidBodies, sizeof(numRigidBodies)); //can be defined outside of the kernel, and only once
	ApplyPeriodicBoundaryXKernel_RigidBodies<<<nBlock_NumRigids, nThreads_RigidBodies>>>(R3CAST(posRigidD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");
	ApplyPeriodicBoundaryYKernel_RigidBodies<<<nBlock_NumRigids, nThreads_RigidBodies>>>(R3CAST(posRigidD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");
	ApplyPeriodicBoundaryZKernel_RigidBodies<<<nBlock_NumRigids, nThreads_RigidBodies>>>(R3CAST(posRigidD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");
}
//--------------------------------------------------------------------------------------------------------------------------------
void FindPassesFromTheEnd(
		thrust::device_vector<real3> & posRigidD,
		thrust::device_vector<int> & distributionD,
		int numRigidBodies,
		real2 pipeCenter,
		real_ pipeRadius,
		int numberOfSections) {
//	real3 posRigid = posRigidD[0];
//	printf("xRigid %f\n", posRadRigid.x);cutil_math deprecate
	real_ dR = pipeRadius / numberOfSections;
	thrust::device_vector<uint> radialPositions(numRigidBodies);
	thrust::device_vector<uint> radialPosCounter(numRigidBodies);
	thrust::fill(radialPositions.begin(), radialPositions.end(), 10000); //10000 as a large number
	thrust::fill(radialPosCounter.begin(), radialPosCounter.end(), 0);

	uint nBlock_NumRigids, nThreads_RigidBodies;
	computeGridSize(numRigidBodies, 128, nBlock_NumRigids, nThreads_RigidBodies);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(numRigidBodiesD), &numRigidBodies, sizeof(numRigidBodies)); //can be defined outside of the kernel, and only once
	PassesFromTheEnd_Kernel<<<nBlock_NumRigids, nThreads_RigidBodies>>>(R3CAST(posRigidD), U1CAST(radialPositions), U1CAST(radialPosCounter), pipeCenter, dR);
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: PassesFromTheEnd_Kernel");

	thrust::sort_by_key(radialPositions.begin(), radialPositions.end(), radialPosCounter.begin());
	thrust::device_vector<uint> radialPosCounter_Cumulative(numberOfSections + 2); //+2 for safety, specially when the particle goes outside of the pipe
	thrust::device_vector<uint> dummy_radialPosition(numberOfSections + 2);
	(void) thrust::reduce_by_key(radialPositions.begin(), radialPositions.end(), radialPosCounter.begin(), dummy_radialPosition.begin(),
			radialPosCounter_Cumulative.begin());
//	radialPosCounter_Cumulative.resize(numberOfSections);
//	dummy_radialPosition.resize(numberOfSections);

	//printf("%$%$%$%$%$%$ dummy_radialPosition[0] %d")

	uint nBlock_NumSections, nThreads_numSections;
	computeGridSize(numberOfSections, 128, nBlock_NumSections, nThreads_numSections);
	AddToCumulutaiveNumberOfPasses<<<nBlock_NumSections, nThreads_numSections>>>(I1CAST(distributionD), U1CAST(dummy_radialPosition), U1CAST(radialPosCounter_Cumulative), numberOfSections);
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: AddToCumulutaiveNumberOfPasses");

	radialPosCounter_Cumulative.clear();
	dummy_radialPosition.clear();
	radialPositions.clear();
	radialPosCounter.clear();
}
//--------------------------------------------------------------------------------------------------------------------------------
void UpdateRigidBody(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real3> & posRigidD,
		thrust::device_vector<real3> & posRigidCumulativeD,
		thrust::device_vector<real4> & velMassRigidD,
		thrust::device_vector<real4> & qD,
		thrust::device_vector<real3> & AD1,
		thrust::device_vector<real3> & AD2,
		thrust::device_vector<real3> & AD3,
		thrust::device_vector<real3> & omegaLRF_D,
		thrust::device_vector<real4> & derivVelRhoD,
		const thrust::device_vector<int> & rigidIdentifierD,
		const thrust::device_vector<real3> & rigidSPH_MeshPos_LRF_D,
		const thrust::host_vector<int3> & referenceArray,
		const thrust::device_vector<real3> & jD1,
		const thrust::device_vector<real3> & jD2,
		const thrust::device_vector<real3> & jInvD1,
		const thrust::device_vector<real3> & jInvD2,
		SimParams paramsH,
		int numRigidBodies,
		int startRigidMarkers,
		int numRigid_SphMarkers,
		float fracSimulation,
		real_ dT) {
	if (referenceArray.size() < 3) {
		return;
	}
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dTD), &dT, sizeof(dT));
//g
	thrust::device_vector<real4> totalSurfaceInteractionRigid4(numRigidBodies);
	thrust::device_vector<real3> totalTorque3(numRigidBodies);
	thrust::fill(totalSurfaceInteractionRigid4.begin(), totalSurfaceInteractionRigid4.end(), R4(0));
	thrust::device_vector<int> dummyIdentify(numRigidBodies);
	thrust::equal_to<int> binary_pred;

	(void) thrust::reduce_by_key(rigidIdentifierD.begin(), rigidIdentifierD.end(), derivVelRhoD.begin() + startRigidMarkers, dummyIdentify.begin(),
			totalSurfaceInteractionRigid4.begin(), binary_pred, thrust::plus<real4>());

	uint nBlocks_numRigid_SphMarkers;
	uint nThreads_SphMarkers;
	computeGridSize(numRigid_SphMarkers, 256, nBlocks_numRigid_SphMarkers, nThreads_SphMarkers);

	thrust::device_vector<real3> totalForcesRigid3(numRigidBodies);
	thrust::fill(totalForcesRigid3.begin(), totalForcesRigid3.end(), R3(0));
	SumSurfaceInteractionForces<<<nBlocks_numRigid_SphMarkers, nThreads_SphMarkers>>>(R3CAST(totalForcesRigid3), R4CAST(totalSurfaceInteractionRigid4), R4CAST(velMassRigidD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: SumSurfaceInteractionForces");
	totalSurfaceInteractionRigid4.clear();



	thrust::device_vector<real3> torqueMarkersD(numRigid_SphMarkers);
	CalcTorqueShare<<<nBlocks_numRigid_SphMarkers, nThreads_SphMarkers>>>(R3CAST(torqueMarkersD), R4CAST(derivVelRhoD), R3CAST(posRadD), I1CAST(rigidIdentifierD), R3CAST(posRigidD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: CalcTorqueShare");
	(void) thrust::reduce_by_key(rigidIdentifierD.begin(), rigidIdentifierD.end(), torqueMarkersD.begin(), dummyIdentify.begin(),
			totalTorque3.begin(), binary_pred, thrust::plus<real3>());

	torqueMarkersD.clear();
	dummyIdentify.clear();

	//add gravity
	thrust::device_vector<real3> gravityForces3(numRigidBodies);
	thrust::fill(gravityForces3.begin(), gravityForces3.end(), paramsH.gravity);
	thrust::transform(totalForcesRigid3.begin(), totalForcesRigid3.end(), gravityForces3.begin(), totalForcesRigid3.begin(), thrust::plus<real3>());
	gravityForces3.clear();

	//################################################### update rigid body things
	uint nBlock_UpdateRigid;
	uint nThreads_rigidParticles;
	computeGridSize(numRigidBodies, 128, nBlock_UpdateRigid, nThreads_rigidParticles);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(numRigidBodiesD), &numRigidBodies, sizeof(numRigidBodies)); //can be defined outside of the kernel, and only once

	// copy solid_SPH_mass to symbol -constant memory
	thrust::device_vector<real3> LF_totalTorque3(numRigidBodies);
	MapTorqueToLRFKernel<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R3CAST(AD1), R3CAST(AD2), R3CAST(AD3), R3CAST(totalTorque3), R3CAST(LF_totalTorque3));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: MapTorqueToLRFKernel");
	totalTorque3.clear();

	if (fracSimulation <.01) {
		UpdateKernelRigidTranstalationBeta<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R3CAST(totalForcesRigid3), R3CAST(posRigidD), R3CAST(posRigidCumulativeD), R4CAST(velMassRigidD));
	} else {
		UpdateKernelRigidTranstalation<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R3CAST(totalForcesRigid3), R3CAST(posRigidD), R3CAST(posRigidCumulativeD), R4CAST(velMassRigidD));
	}
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");
	totalForcesRigid3.clear();

	UpdateRigidBodyQuaternion_kernel<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R4CAST(qD), R3CAST(omegaLRF_D));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateRotation");

	RotationMatirixFromQuaternion_kernel<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R3CAST(AD1), R3CAST(AD2), R3CAST(AD3), R4CAST(qD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateRotation");

	UpdateRigidBodyAngularVelocity_kernel<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R3CAST(LF_totalTorque3), R3CAST(jD1), R3CAST(jD2), R3CAST(jInvD1), R3CAST(jInvD2), R3CAST(omegaLRF_D));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");

	LF_totalTorque3.clear();
	//################################################### update rigid body things
	UpdateRigidMarkersPosition<<<nBlocks_numRigid_SphMarkers, nThreads_SphMarkers>>>(R3CAST(posRadD), R4CAST(velMasD), R3CAST(rigidSPH_MeshPos_LRF_D), I1CAST(rigidIdentifierD), R3CAST(posRigidD), R4CAST(velMassRigidD), R3CAST(omegaLRF_D), R3CAST(AD1), R3CAST(AD2), R3CAST(AD3));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");
}
//--------------------------------------------------------------------------------------------------------------------------------
void UpdateFlexibleBody(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real4> & derivVelRhoD,
		const int numRigidBodies,
		const int numFlexBodies,
		const int numFlex_SphMarkers,
		thrust::device_vector<real3> & ANCF_NodesD,
		thrust::device_vector<real3> & ANCF_SlopesD,
		thrust::device_vector<real3> & ANCF_NodesVelD,
		thrust::device_vector<real3> & ANCF_SlopesVelD,
		thrust::device_vector<int2> & ANCF_ReferenceArrayNodesOnBeamsD,
		thrust::device_vector<int> & ANCF_NumMarkers_Per_BeamD,
		thrust::device_vector<int> & ANCF_NumMarkers_Per_Beam_CumulD,
//		thrust::device_vector<int> & ANCF_NumNodesMultMarkers_Per_BeamD,
		thrust::device_vector<int> & ANCF_NumNodesMultMarkers_Per_Beam_CumulD,

		const thrust::device_vector<int> & flexIdentifierD,
		const thrust::device_vector<int2> & flexMapEachMarkerOnAllBeamNodesD,
		const thrust::device_vector<real3> & flexSPH_MeshPos_LRF_D,
		const thrust::device_vector<real_> & flexParametricDistD,
		const thrust::device_vector<real_> & ANCF_Beam_LengthD,
		const thrust::host_vector<int3> & referenceArray,

		SimParams paramsH,
		float fracSimulation,
		real_ dT) {
	if (numFlexBodies == 0) {
		return;
	}
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dTD), &dT, sizeof(dT));

	int numFlBcRigid = 2 + numRigidBodies;
	int totalNumberOfFlexNodes = ANCF_ReferenceArrayNodesOnBeamsD[ANCF_ReferenceArrayNodesOnBeamsD.size() - 1].y;
	int totalNumberOfFlexMultNodes = flexMapEachMarkerOnAllBeamNodesD.size();

	thrust::device_vector<real3> flex_FSI_NodesForces1(totalNumberOfFlexNodes);
	thrust::device_vector<real3> flex_FSI_NodesForces2(totalNumberOfFlexNodes);
	thrust::fill(flex_FSI_NodesForces1.begin(), flex_FSI_NodesForces1.end(),0);
	thrust::fill(flex_FSI_NodesForces2.begin(), flex_FSI_NodesForces2.end(),0);

	thrust::device_vector<real3> flexNodesForcesAllMarkers1(totalNumberOfFlexMultNodes);
	thrust::device_vector<real3> flexNodesForcesAllMarkers2(totalNumberOfFlexMultNodes);

	uint nBlocks_numFlex_SphMarkers;
	uint nThreads_SphMarkers;
	computeGridSize(numFlex_SphMarkers, 256, nBlocks_numFlex_SphMarkers, nThreads_SphMarkers);
	MapForcesOnNodes<<<nBlocks_numFlex_SphMarkers, nThreads_SphMarkers>>>(
			R3CAST(flexNodesForcesAllMarkers1),
			R3CAST(flexNodesForcesAllMarkers2),
			I1CAST(flexIdentifierD),
			I1CAST(ANCF_NumMarkers_Per_BeamD),
			I1CAST(ANCF_NumMarkers_Per_Beam_CumulD),
			I1CAST(ANCF_NumNodesMultMarkers_Per_Beam_CumulD),
			R1CAST(flexParametricDistD),
			R4CAST(derivVelRhoD));

	if (flexMapEachMarkerOnAllBeamNodesD.size() != flexNodesForcesAllMarkers1.size()) {
		printf("we have size inconsistency between flex nodesForces and nodesPair identifier");
	}
	thrust::device_vector<int> dummyNodesFlexIdentify(flexMapEachMarkerOnAllBeamNodesD.size());
	thrust::equal_to<int2> binary_pred_int2; //if binary_pred int2 does not work, you have to either add operator == to custom_cutil_math, or you have to map nodes identifiers from int2 to int
	(void) thrust::reduce_by_key(flexMapEachMarkerOnAllBeamNodesD.begin(), flexMapEachMarkerOnAllBeamNodesD.end(), flexNodesForcesAllMarkers1.begin(), dummyNodesFlexIdentify.begin(),
			flex_FSI_NodesForces1.begin(), binary_pred_int2, thrust::plus<real3>());
	(void) thrust::reduce_by_key(flexMapEachMarkerOnAllBeamNodesD.begin(), flexMapEachMarkerOnAllBeamNodesD.end(), flexNodesForcesAllMarkers2.begin(), dummyNodesFlexIdentify.begin(),
			flex_FSI_NodesForces2.begin(), binary_pred_int2, thrust::plus<real3>());
	flexNodesForcesAllMarkers1.clear();
	flexNodesForcesAllMarkers2.clear();

//	//TODO: update flex bodies here
	Update_ANCF_Beam(
			ANCF_NodesD, ANCF_SlopesD, ANCF_NodesVelD, ANCF_SlopesVelD,
			flex_FSI_NodesForces1, flex_FSI_NodesForces2,
			ANCF_ReferenceArrayNodesOnBeamsD, ANCF_Beam_LengthD,
			numFlexBodies, dT
			);


//	 ....
//	 ....
//	 ....
//	 ....
//	//end

//	//TODO: add gravity to Flex objects
//	thrust::device_vector<real3> gravityForces3(numRigidBodies);
//	thrust::fill(gravityForces3.begin(), gravityForces3.end(), paramsH.gravity);
//	thrust::transform(totalForcesRigid3.begin(), totalForcesRigid3.end(), gravityForces3.begin(), totalForcesRigid3.begin(), thrust::plus<real3>());
//	gravityForces3.clear();
//	//

	//################################################### update rigid body things
	computeGridSize(numFlex_SphMarkers, 256, nBlocks_numFlex_SphMarkers, nThreads_SphMarkers);

	UpdateFlexMarkersPosition<<<nBlocks_numFlex_SphMarkers, nThreads_SphMarkers>>>(
			R3CAST(posRadD), R4CAST(velMasD),
			I1CAST(flexIdentifierD),
			R3CAST(flexSPH_MeshPos_LRF_D),
			R1CAST(flexParametricDistD),
			R1CAST(ANCF_Beam_LengthD),
			I2CAST(ANCF_ReferenceArrayNodesOnBeamsD),
			R3CAST(ANCF_NodesD),
			R3CAST(ANCF_SlopesD),
			R3CAST(ANCF_NodesVelD),
			R3CAST(ANCF_SlopesVelD)
			);

	hipDeviceSynchronize();

	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");


	//------------------------ delete stuff
	dummyNodesFlexIdentify.clear();


	flex_FSI_NodesForces1.clear();
	flex_FSI_NodesForces2.clear();
}
//##############################################################################################################################################
// the main function, which updates the particles and implements BC
void cudaCollisions(
		thrust::host_vector<real3> & mPosRad,
		thrust::host_vector<real4> & mVelMas,
		thrust::host_vector<real4> & mRhoPresMu,
		const thrust::host_vector<uint> & bodyIndex,
		const thrust::host_vector<int3> & referenceArray,

		const thrust::host_vector<real3> & ANCF_Nodes,
		const thrust::host_vector<real3> & ANCF_Slopes,
		const thrust::host_vector<real3> & ANCF_NodesVel,
		const thrust::host_vector<real3> & ANCF_SlopesVel,
		const thrust::host_vector<real_> & ANCF_Beam_Length,
		const thrust::host_vector<int2> & ANCF_ReferenceArrayNodesOnBeams,

		const thrust::host_vector<real_> & flexParametricDist,
		int & numAllMarkers,
		real3 cMax,
		real3 cMin,
		real_ delT,
		thrust::host_vector<real3> & posRigidH,
		thrust::host_vector<real4> & mQuatRot,
		thrust::host_vector<real4> & velMassRigidH,
		thrust::host_vector<real3> omegaLRF_H,
		thrust::host_vector<real3> jH1,
		thrust::host_vector<real3> jH2,
		thrust::host_vector<real3> jInvH1,
		thrust::host_vector<real3> jInvH2,
		real_ binSize0,
		real_ channelRadius,
		real2 channelCenterYZ) {
	//--------- initialization ---------------
	//hipError_t dumDevErr = hipSetDevice(2);
	GpuTimer myTotalTime;
	myTotalTime.Start();
	printf("a1 yoho\n");
	//printf("cMin.x, y, z, CMAx.x, y, z, binSize %f %f %f , %f %f %f, %f\n", cMin.x, cMin.y, cMin.z, cMax.x, cMax.y, cMax.z, binSize0); 
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	hipMemcpyToSymbolAsync(HIP_SYMBOL(cMinD), &cMin, sizeof(cMin));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(cMaxD), &cMax, sizeof(cMax));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(numAllMarkersD), &numAllMarkers, sizeof(numAllMarkers));
	printf("a2 yoho\n");

	int numRigidBodies = posRigidH.size();
	thrust::device_vector<real3> posRadD=mPosRad;
	//thrust::copy(mPosRad.begin(), mPosRad.end(), posRadD.begin());
	thrust::device_vector<real4> velMasD=mVelMas;
	//thrust::copy(mVelMas.begin(), mVelMas.end(), velMasD.begin());
	thrust::device_vector<real4> rhoPresMuD=mRhoPresMu;
	//thrust::copy(mRhoPresMu.begin(), mRhoPresMu.end(), rhoPresMuD.begin());
	printf("a3 yoho\n");

	thrust::device_vector<real3> posRigidD=posRigidH;
	//thrust::copy(posRigidH.begin(), posRigidH.end(), posRigidD.begin());
	thrust::device_vector<real3> posRigidCumulativeD=posRigidH;
	//thrust::copy(posRigidH.begin(), posRigidH.end(), posRigidCumulativeD.begin());
	thrust::device_vector<real4> velMassRigidD=velMassRigidH;
	//thrust::copy(velMassRigidH.begin(), velMassRigidH.end(), velMassRigidD.begin());
	thrust::device_vector<real3> omegaLRF_D=omegaLRF_H;
	//thrust::copy(omegaLRF_H.begin(), omegaLRF_H.end(), omegaLRF_D.begin());
	printf("a4 yoho\n");
	thrust::device_vector<real3> jD1=jH1;
	thrust::device_vector<real3> jD2=jH2;
	thrust::device_vector<real3> jInvD1=jInvH1;
	thrust::device_vector<real3> jInvD2=jInvH2;
	//thrust::copy(jH1.begin(), jH1.end(), jD1.begin());
	//thrust::copy(jH2.begin(), jH2.end(), jD2.begin());
	//thrust::copy(jInvH1.begin(), jInvH1.end(), jInvD1.begin());
	//thrust::copy(jInvH2.begin(), jInvH2.end(), jInvD2.begin());
	printf("a5 yoho\n");
	thrust::device_vector<uint> bodyIndexD=bodyIndex;
	//thrust::copy(bodyIndex.begin(), bodyIndex.end(), bodyIndexD.begin());
	thrust::device_vector<real4> derivVelRhoD(numAllMarkers);
	printf("a6 yoho\n");
		//******************** rigid body some initialization
	real_ solid_SPH_mass;																					//____________________________> typical mass, save to constant memory
	int numRigid_SphMarkers = 0;
	int startRigidMarkers = (referenceArray[1]).y;
	numRigid_SphMarkers = referenceArray[2 + numRigidBodies - 1].y - startRigidMarkers;
	thrust::device_vector<int> rigidIdentifierD(numRigid_SphMarkers);
	if (numRigidBodies > 0) {
		real4 typicalRigidSPH = mVelMas[referenceArray[2].x];
		solid_SPH_mass = typicalRigidSPH.w;
	} else {
		real4 dummyFluid = mVelMas[referenceArray[0].x];
		solid_SPH_mass = 100 * dummyFluid.w;
	}
	cutilSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(solid_SPH_massD), &solid_SPH_mass, sizeof(solid_SPH_mass)));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(startRigidMarkersD), &startRigidMarkers, sizeof(startRigidMarkers)); //can be defined outside of the kernel, and only once
	hipMemcpyToSymbolAsync(HIP_SYMBOL(numRigid_SphMarkersD), &numRigid_SphMarkers, sizeof(numRigid_SphMarkers)); //can be defined outside of the kernel, and only once

	MakeRigidIdentifier(rigidIdentifierD, numRigidBodies, startRigidMarkers, referenceArray);

	printf("a7 yoho\n");

		//******************************************************************************
	thrust::device_vector<real3> rigidSPH_MeshPos_LRF_D(numRigid_SphMarkers);
	uint nBlocks_numRigid_SphMarkers;
	uint nThreads_SphMarkers;
	computeGridSize(numRigid_SphMarkers, 256, nBlocks_numRigid_SphMarkers, nThreads_SphMarkers);
	printf("before first kernel\n");
	Populate_RigidSPH_MeshPos_LRF_kernel<<<nBlocks_numRigid_SphMarkers, nThreads_SphMarkers>>>(R3CAST(rigidSPH_MeshPos_LRF_D), R3CAST(posRadD), I1CAST(rigidIdentifierD), R3CAST(posRigidD), startRigidMarkers, numRigid_SphMarkers);
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: CalcTorqueShare");	printf("after first kernel\n");

	//******************************************************************************
	//******************** flex body some initialization

	int numFlBcRigid = 2 + numRigidBodies;
	int numFlexBodies = ANCF_Beam_Length.size();
//	int totalNumberOfFlexNodes = ANCF_ReferenceArrayNodesOnBeamsD[ANCF_ReferenceArrayNodesOnBeamsD.size() - 1].y;

	int startFlexMarkers = (referenceArray[numFlBcRigid-1]).y;
	int numFlex_SphMarkers = referenceArray[numFlBcRigid + numFlexBodies - 1].y - startFlexMarkers;

	hipMemcpyToSymbolAsync(HIP_SYMBOL(startFlexMarkersD), &startFlexMarkers, sizeof(startFlexMarkers)); //can be defined outside of the kernel, and only once
	hipMemcpyToSymbolAsync(HIP_SYMBOL(numFlex_SphMarkersD), &numFlex_SphMarkers, sizeof(numFlex_SphMarkers)); //can be defined outside of the kernel, and only once

	//******************************************************************************
	thrust::device_vector<real_> flexParametricDistD = flexParametricDist;
	thrust::device_vector<int> flexIdentifierD(numFlex_SphMarkers);

	MakeFlexIdentifier(flexIdentifierD, numFlexBodies, numFlBcRigid, startFlexMarkers, referenceArray);

	thrust::device_vector<real3> ANCF_NodesD = ANCF_Nodes;
	thrust::device_vector<real3> ANCF_SlopesD = ANCF_Slopes;
	thrust::device_vector<real3> ANCF_NodesVelD = ANCF_NodesVel;
	thrust::device_vector<real3> ANCF_SlopesVelD = ANCF_SlopesVel;
	thrust::device_vector<real_> ANCF_Beam_LengthD = ANCF_Beam_Length;
	thrust::device_vector<int2> ANCF_ReferenceArrayNodesOnBeamsD = ANCF_ReferenceArrayNodesOnBeams;  //each element refer to a beam. and contains the start and end
																									// index of nodes in the array of beams nodes (like referenceArray)

	//*******************
	thrust::device_vector<int> ANCF_NumMarkers_Per_BeamD(numFlexBodies);  //num BCE markers per beam
	thrust::device_vector<int> ANCF_NumMarkers_Per_Beam_CumulD(numFlexBodies); // exclusive scan of ANCF_NumMarkers_Per_BeamD
	thrust::device_vector<int> ANCF_NumNodesMultMarkers_Per_BeamD(numFlexBodies); //i_th component is equal to nN*nM (N and M denote nodes and markers per beam) of beam i
	thrust::device_vector<int> ANCF_NumNodesMultMarkers_Per_Beam_CumulD(numFlexBodies); //exclusive scan of ANCF_NumNodesMultMarkers_Per_BeamD
	thrust::device_vector<int2> flexMapEachMarkerOnAllBeamNodesD(0); //assume beam i has nN nodes and nM markers. lets j denote the nodes. This array includes
																	// concequtive chunks of pairs I2(i,j). Each chunk has a length of nM. The total number of chunks
																	// per beam is nN. In summary, nN chuncks of I2(i, j) pairs (j changes from 0 to nN), Each chunk with
																	// with the length of nM

	thrust::device_vector<int> dummySum(flexIdentifierD.size());
	thrust::device_vector<int> dummyIdentifier(0);
	thrust::fill(dummySum.begin(), dummySum.end(), 1);
	(void) thrust::reduce_by_key(flexIdentifierD.begin(), flexIdentifierD.end(), dummySum.begin(), dummyIdentifier.begin(), ANCF_NumMarkers_Per_BeamD.begin());
	thrust::exclusive_scan(ANCF_NumMarkers_Per_BeamD.begin(), ANCF_NumMarkers_Per_BeamD.end(), ANCF_NumMarkers_Per_Beam_CumulD());
	dummySum.clear();
	dummyIdentifier.clear();

	Calc_NumNodesMultMarkers_Per_Beam(ANCF_NumNodesMultMarkers_Per_BeamD, ANCF_NumMarkers_Per_BeamD, ANCF_ReferenceArrayNodesOnBeams, numFlexBodies);
	thrust::exclusive_scan(ANCF_NumNodesMultMarkers_Per_BeamD.begin(), ANCF_NumNodesMultMarkers_Per_BeamD.end(), ANCF_NumNodesMultMarkers_Per_Beam_CumulD());

	Calc_mapEachMarkerOnAllBeamNodes_IdentifierD(flexMapEachMarkerOnAllBeamNodesD, ANCF_NumNodesMultMarkers_Per_Beam_CumulD, ANCF_NumMarkers_Per_BeamD, ANCF_ReferenceArrayNodesOnBeams, numFlexBodies);

	//*******************

	thrust::device_vector<real3> flexSPH_MeshPos_LRF_D(numFlex_SphMarkers);
	thrust::device_vector<real3> flexSPH_MeshSlope_Initial_D(numFlex_SphMarkers);  //slope of the beam at BCE marker (associated to BCE marker)
	uint nBlocks_numFlex_SphMarkers;
	uint nThreads_SphMarkers;
	computeGridSize(numFlex_SphMarkers, 256, nBlocks_numFlex_SphMarkers, nThreads_SphMarkers);
	printf("before first kernel\n");

	Populate_FlexSPH_MeshPos_LRF_kernel<<<nBlocks_numFlex_SphMarkers, nThreads_SphMarkers>>>(R3CAST(flexSPH_MeshPos_LRF_D), R3CAST(posRadD), I1CAST(flexIdentifierD), R1CAST(flexParametricDistD), R1CAST(ANCF_Beam_LengthD),
			I2CAST(ANCF_ReferenceArrayNodesOnBeamsD), R3CAST(ANCF_NodesD), R3CAST(ANCF_SlopesD));
	hipDeviceSynchronize();
		CUT_CHECK_ERROR("Kernel execution failed: Populate_FlexSPH_MeshPos_LRF_kernel");	printf("after first kernel\n");

	Populate_FlexSPH_MeshSlope_LRF_kernel<<<nBlocks_numFlex_SphMarkers, nThreads_SphMarkers>>>(R3CAST(flexSPH_MeshSlope_Initial_D), I1CAST(flexIdentifierD), R1CAST(flexParametricDistD), R1CAST(ANCF_Beam_LengthD),
				I2CAST(ANCF_ReferenceArrayNodesOnBeamsD), R3CAST(ANCF_NodesD), R3CAST(ANCF_SlopesD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: Populate_FlexSPH_MeshSlope_LRF_kernel");	printf("after first kernel\n");

	//******************************************************************************
	thrust::device_vector<real4> qD1 = mQuatRot;
	thrust::device_vector<real3> AD1(numRigidBodies);
	thrust::device_vector<real3> AD2(numRigidBodies);
	thrust::device_vector<real3> AD3(numRigidBodies);
	uint nBlock_UpdateRigid;
	uint nThreads_rigidParticles;
	computeGridSize(numRigidBodies, 128, nBlock_UpdateRigid, nThreads_rigidParticles);
	RotationMatirixFromQuaternion_kernel<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R3CAST(AD1), R3CAST(AD2), R3CAST(AD3), R4CAST(qD1));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateRotation");

	//int i =  rigidIdentifierD[429];
	//printf("rigid body coord %d %f %f\n", i, posRigidH[i].x, posRigidH[i].z);
	//printf("length %f\n", length(R2(posRigidH[i].x - .003474, posRigidH[i].z - .000673)));

	//****************************** bin size adjustement and contact detection stuff *****************************
	//real_ mBinSize0 = (numAllMarkers == 0) ? mBinSize0 : 2 * HSML;
	//real3 cMinOffsetCollisionPurpose = cMin - 3 * R3(0, mBinSize0, mBinSize0);		//periodic bc in x direction
	//real3 cMaxOffsetCollisionPurpose = cMax + 3 * R3(0, mBinSize0, mBinSize0);
	////real3 cMinOffsetCollisionPurpose = cMin - 3 * R3(mBinSize0, mBinSize0, mBinSize0);		//periodic bc in x direction
	////real3 cMaxOffsetCollisionPurpose = cMax + 3 * R3(mBinSize0, mBinSize0, mBinSize0);

	/////printf("side.x %f\n", abs(cMaxOffsetCollisionPurpose.x - cMinOffsetCollisionPurpose.x) / mBinSize);
	//int3 SIDE = I3(  floor( (cMaxOffsetCollisionPurpose.x - cMinOffsetCollisionPurpose.x) / mBinSize0 ), floor( (cMaxOffsetCollisionPurpose.y - cMinOffsetCollisionPurpose.y) / mBinSize0 ), floor( (cMaxOffsetCollisionPurpose.z - cMinOffsetCollisionPurpose.z) / mBinSize0)  );
	//real_ mBinSize = (cMaxOffsetCollisionPurpose.x - cMinOffsetCollisionPurpose.x) / SIDE.x;  //this one works when periodic BC is only on x. if it was on y as well (or on z), you would have problem.
	real3 cMinOffsetCollisionPurpose = cMin;// - 3 * R3(0, 0, binSize0); //periodic bc in x direction
	real3 cMaxOffsetCollisionPurpose = cMax;// + 3 * R3(0, 0, binSize0);
	int3 SIDE = I3(int((cMaxOffsetCollisionPurpose.x - cMinOffsetCollisionPurpose.x) / binSize0 + .1), int((cMaxOffsetCollisionPurpose.y - cMinOffsetCollisionPurpose.y) / binSize0 + .1),
			floor((cMaxOffsetCollisionPurpose.z - cMinOffsetCollisionPurpose.z) / binSize0 + .1));
	real_ mBinSize = binSize0; //Best solution in that case may be to change cMax or cMin such that periodic sides be a multiple of binSize

	printf("SIDE: %d, %d, %d\n", SIDE.x, SIDE.y, SIDE.z);
	//*******************
	SimParams paramsH;
	paramsH.gravity = Gravity; //Gravity * sizeScale;;// R3(0, -9.8, 0) * sizeScale; //R3(0, -9800, 0) * sizeScale;
	paramsH.markerRadius = HSML;
	paramsH.gridSize = SIDE;
	//paramsH.numCells = SIDE.x * SIDE.y * SIDE.z;
	paramsH.worldOrigin = cMinOffsetCollisionPurpose;
	paramsH.cellSize = R3(mBinSize, mBinSize, mBinSize);
	paramsH.boxDims = cMaxOffsetCollisionPurpose - cMinOffsetCollisionPurpose;
	printf("boxDims: %f, %f, %f\n", paramsH.boxDims.x, paramsH.boxDims.y, paramsH.boxDims.z);

	setParameters(&paramsH);
	cutilSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), &paramsH, sizeof(SimParams)));

	//********************************************************************************
	int numberOfSections = 20; //number of sections for measuring the distribution
	thrust::device_vector<int>  distributionD(numberOfSections);

	FILE *outFileMultipleZones;

	int povRayCounter = 0;
	int stepEnd = 1.0e6;//2.4e6;//600000;//2.4e6 * (.02 * sizeScale) / delT ; //1.4e6 * (.02 * sizeScale) / delT ;//0.7e6 * (.02 * sizeScale) / delT ;//0.7e6;//2.5e6; //200000;//10000;//50000;//100000;
	printf("stepEnd %d\n", stepEnd);

	real_ delTOrig = delT;
	//for (int tStep = 0; tStep < 0; tStep ++) {
	for (int tStep = 0; tStep < stepEnd + 1; tStep++) {
		//edit  since yu deleted cyliderRotOmegaJD

//		if (tStep > 10000) delT = .2;
		GpuTimer myGpuTimer;
		myGpuTimer.Start();

		if (tStep < 1000) delT = 0.25 * delTOrig; else delT = delTOrig;
		//computations
		thrust::device_vector<real3> posRadD2 = posRadD;
		thrust::device_vector<real4> velMasD2 = velMasD;
		thrust::device_vector<real4> rhoPresMuD2 = rhoPresMuD;
		thrust::device_vector<real3> posRigidD2 = posRigidD;
		thrust::device_vector<real3> posRadRigidCumulativeD2 = posRigidCumulativeD;
		thrust::device_vector<real4> velMassRigidD2 = velMassRigidD;
		thrust::device_vector<real3> omegaLRF_D2 = omegaLRF_D;
		thrust::device_vector<real3> vel_XSPH_D(numAllMarkers);
		thrust::device_vector<real3> AD1_2 = AD1;
		thrust::device_vector<real3> AD2_2 = AD2;
		thrust::device_vector<real3> AD3_2 = AD3;
		thrust::device_vector<real4> qD2 = qD1;

		//******** RK2
		ForceSPH(posRadD, velMasD, vel_XSPH_D, rhoPresMuD, bodyIndexD, derivVelRhoD, referenceArray, numAllMarkers, SIDE, 0.5 * delT); //?$ right now, it does not consider gravity or other stuff on rigid bodies. they should be applied at rigid body solver
		UpdateFluid(posRadD2, velMasD2, vel_XSPH_D, rhoPresMuD2, derivVelRhoD, referenceArray, 0.5 * delT); //assumes ...D2 is a copy of ...D
		//UpdateBoundary(posRadD2, velMasD2, rhoPresMuD2, derivVelRhoD, referenceArray, 0.5 * delT);		//assumes ...D2 is a copy of ...D
		UpdateRigidBody(posRadD2, velMasD2, posRigidD2, posRadRigidCumulativeD2, velMassRigidD2, qD2, AD1_2, AD2_2, AD3_2, omegaLRF_D2, derivVelRhoD, rigidIdentifierD,
				rigidSPH_MeshPos_LRF_D, referenceArray, jD1, jD2, jInvD1, jInvD2, paramsH, numRigidBodies, startRigidMarkers, numRigid_SphMarkers, float(tStep)/stepEnd, 0.5 * delT);
		// UpdateFlexibleBody
		ApplyBoundary(posRadD2, rhoPresMuD2, numAllMarkers, posRigidD2, velMassRigidD2, numRigidBodies);
		//*****
		ForceSPH(posRadD2, velMasD2, vel_XSPH_D, rhoPresMuD2, bodyIndexD, derivVelRhoD, referenceArray, numAllMarkers, SIDE, delT);
		UpdateFluid(posRadD, velMasD, vel_XSPH_D, rhoPresMuD, derivVelRhoD, referenceArray, delT);
		//UpdateBoundary(posRadD, velMasD, rhoPresMuD, derivVelRhoD, referenceArray, delT);
		UpdateRigidBody(posRadD, velMasD, posRigidD, posRigidCumulativeD, velMassRigidD, qD1, AD1, AD2, AD3, omegaLRF_D, derivVelRhoD, rigidIdentifierD,
				rigidSPH_MeshPos_LRF_D, referenceArray, jD1, jD2, jInvD1, jInvD2, paramsH, numRigidBodies, startRigidMarkers, numRigid_SphMarkers, float(tStep)/stepEnd, delT);
		// UpdateFlexibleBody
		UpdateFlexibleBody(posRadD, velMasD, derivVelRhoD,
						numRigidBodies, numFlexBodies, numFlex_SphMarkers,
						ANCF_NodesD, ANCF_SlopesD, ANCF_NodesVelD, ANCF_SlopesVelD,
						ANCF_ReferenceArrayNodesOnBeamsD,
						ANCF_NumMarkers_Per_BeamD,
						ANCF_NumMarkers_Per_Beam_CumulD,
				//		thrust::device_vector<int> & ANCF_NumNodesMultMarkers_Per_BeamD,
						ANCF_NumNodesMultMarkers_Per_Beam_CumulD,

						flexIdentifierD,
						flexMapEachMarkerOnAllBeamNodesD,
						flexSPH_MeshPos_LRF_D,
						flexParametricDistD,
						ANCF_Beam_LengthD,
						referenceArray,

						paramsH,
						float(tStep)/stepEnd,
						delT);
		ApplyBoundary(posRadD, rhoPresMuD, numAllMarkers, posRigidD, velMassRigidD, numRigidBodies);
		//************




//			/* post_process for Segre-Silberberg */ goes before ApplyBoundary
//			if(tStep >= 0) {
//				real2 channelCenter = .5 * R2(cMax.y + cMin.y, cMax.z + cMin.z);
//				FindPassesFromTheEnd(posRigidD, distributionD, numRigidBodies, channelCenter, channelRadius, numberOfSections);
//			}



		posRadD2.clear();
		velMasD2.clear();
		rhoPresMuD2.clear();
		posRigidD2.clear();
		posRadRigidCumulativeD2.clear();
		velMassRigidD2.clear();
		vel_XSPH_D.clear();
		qD2.clear();
		AD1_2.clear();
		AD2_2.clear();
		AD3_2.clear();
		omegaLRF_D2.clear();

		//density re-initialization
		if (tStep % 10 == 0) {
			DensityReinitialization(posRadD, velMasD, rhoPresMuD, numAllMarkers, SIDE); //does not work for analytical boundaries (non-meshed) and free surfaces
		}

		//************************************************
		//edit  since yu deleted cyliderRotOmegaJD
		PrintToFile(posRadD, velMasD, rhoPresMuD, referenceArray, rigidIdentifierD, posRigidD, posRigidCumulativeD, velMassRigidD, qD1, AD1, AD2, AD3, omegaLRF_D, cMax, cMin, paramsH,
				delT, tStep, channelRadius, channelCenterYZ);

//		PrintToFileDistribution(distributionD, channelRadius, numberOfSections, tStep);
		//************
		myGpuTimer.Stop();
		real_ time2 = (real_)myGpuTimer.Elapsed();
		if (tStep % 50 == 0) {
			printf("step: %d, step Time: %f\n ", tStep, time2);
			//printf("a \n");
		}
		fflush(stdout);

		//_CrtDumpMemoryLeaks(); //for memory leak detection (msdn suggestion for VS) apparently does not work in conjunction with cuda

	}

	//you may copy back to host
	posRadD.clear();
	velMasD.clear();
	rhoPresMuD.clear();
	posRigidD.clear();

	ANCF_NodesD.clear();
	ANCF_SlopesD.clear();
	ANCF_NodesVelD.clear();
	ANCF_SlopesVelD.clear();
	ANCF_Beam_LengthD.clear();
	ANCF_ReferenceArrayNodesOnBeamsD.clear();

	ANCF_NumMarkers_Per_BeamD.clear();
	ANCF_NumMarkers_Per_Beam_CumulD.clear();
	ANCF_NumNodesMultMarkers_Per_BeamD.clear();
	ANCF_NumNodesMultMarkers_Per_Beam_CumulD.clear();
	flexMapEachMarkerOnAllBeamNodesD.clear();


	posRigidCumulativeD.clear();
	velMassRigidD.clear();
	omegaLRF_D.clear();
	bodyIndexD.clear();
	derivVelRhoD.clear();
	rigidIdentifierD.clear();
	rigidSPH_MeshPos_LRF_D.clear();
	flexParametricDistD.clear();
	flexIdentifierD.clear();
	flexSPH_MeshPos_LRF_D.clear();
	flexSPH_MeshSlope_Initial_D.clear();
	qD1.clear();
	AD1.clear();
	AD2.clear();
	AD3.clear();
	distributionD.clear();

	jD1.clear();
	jD2.clear();
	jInvD1.clear();
	jInvD2.clear();

	myTotalTime.Stop();
	real_ time = (real_)myTotalTime.Elapsed();
	printf("total Time: %f\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n ", time);
}
