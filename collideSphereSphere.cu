#include "hip/hip_runtime.h"
#include "custom_cutil_math.h"
#include "SPHCudaUtils.h"
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "collideSphereSphere.cuh"
#include "SDKCollisionSystem.cuh"
#include "printToFile.cuh"
#include <string.h>
#include <stdio.h>
#include <math.h>
#include <sstream>
#include <fstream>
using namespace std;
//#####################################################################################
#define B_SIZE 128
//#####################################################################################
__constant__ int mNumSpheresD;
__constant__ real_ dTD;
__constant__ real_ rigid_SPH_massD;
__constant__ int2 updatePortionD;
__constant__ real3 cMinD;
__constant__ real3 cMaxD;
__constant__ int2 portionD;
__constant__ int flagD;
__constant__ int numRigidBodiesD;
__constant__ int startRigidMarkersD;
__constant__ int startFlexMarkersD;
__constant__ int numRigid_SphMarkersD;
__constant__ int numFlex_SphMarkersD;

int maxblock = 65535;
//--------------------------------------------------------------------------------------------------------------------------------
//updates the fluid particles' properties, i.e. velocity, density, pressure, position
__global__ void UpdateKernelFluid(real3 * posRadD, real4 * velMasD, real3 * vel_XSPH_D, real4 * rhoPresMuD, real4 * derivVelRhoD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	index += updatePortionD.x; // updatePortionD = [start, end] index of the update portion
	if (index >= updatePortionD.y) {
		return;
	}
	real3 vel_XSPH = vel_XSPH_D[index];
	// 1*** let's tweak a little bit :)
	if (length(vel_XSPH) > .2 * HSML / dTD) {
		vel_XSPH *= ( .2 * HSML / dTD ) / length(vel_XSPH);
	}
	// 1*** end tweak
	real3 posRad = posRadD[index];
	real3 updatedPositon = posRad + vel_XSPH * dTD;
	posRadD[index] = updatedPositon; //posRadD updated

	real4 derivVelRho = derivVelRhoD[index];
	real4 velMas = velMasD[index];
	real3 updatedVelocity = R3(velMas + derivVelRho * dTD);
	// 2*** let's tweak a little bit :)
	if (length(updatedVelocity) > .2 * HSML / dTD) {
		updatedVelocity *= ( .2 * HSML / dTD ) / length(updatedVelocity);
	}
	// 2*** end tweak
	velMasD[index] = R4(updatedVelocity, /*rho2 / rhoPresMu.x * */velMas.w); //velMasD updated

	real4 rhoPresMu = rhoPresMuD[index];
	real_ rho2 = rhoPresMu.x + derivVelRho.w * dTD; //rho update. (i.e. rhoPresMu.x), still not wriiten to global matrix
	rhoPresMu.y = Eos(rho2, rhoPresMu.w);
	rhoPresMu.x = rho2;
	rhoPresMuD[index] = rhoPresMu; //rhoPresMuD updated
}
//--------------------------------------------------------------------------------------------------------------------------------
//copies the sortedVelXSPH to velXSPH according to indexing
__global__ void Copy_SortedVelXSPH_To_VelXSPH(real3 * vel_XSPH_D, real3 * vel_XSPH_Sorted_D, uint * m_dGridParticleIndex, int numParticles) {
	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numParticles) return;
	vel_XSPH_D[m_dGridParticleIndex[index]] = vel_XSPH_Sorted_D[index];
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the fluid particles' properties, i.e. velocity, density, pressure, position
__global__ void UpdateKernelBoundary(real3 * posRadD, real4 * velMasD, real4 * rhoPresMuD, real4 * derivVelRhoD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	index += updatePortionD.x; // updatePortionD = [start, end] index of the update portion
	if (index >= updatePortionD.y) {
		return;
	}

	real4 derivVelRho = derivVelRhoD[index];
	real4 rhoPresMu = rhoPresMuD[index];
	real_ rho2 = rhoPresMu.x + derivVelRho.w * dTD; //rho update. (i.e. rhoPresMu.x), still not wriiten to global matrix
	rhoPresMu.y = Eos(rho2, rhoPresMu.w);
	rhoPresMu.x = rho2;
	rhoPresMuD[index] = rhoPresMu; //rhoPresMuD updated
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along x
__global__ void ApplyPeriodicBoundaryXKernel(real3 * posRadD, real4 * rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= mNumSpheresD) {
		return;
	}
	real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	} //no need to do anything if it is a boundary particle
	real3 posRad = posRadD[index];
	if (posRad.x > cMaxD.x) {
		posRad.x -= (cMaxD.x - cMinD.x);
		posRadD[index] = posRad;
		return;
	}
	if (posRad.x < cMinD.x) {
		posRad.x += (cMaxD.x - cMinD.x);
		posRadD[index] = posRad;
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along y
__global__ void ApplyPeriodicBoundaryYKernel(real3 * posRadD, real4 * rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= mNumSpheresD) {
		return;
	}
	real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	} //no need to do anything if it is a boundary particle
	real3 posRad = posRadD[index];
	if (posRad.y > cMaxD.y) {
		posRad.y -= (cMaxD.y - cMinD.y);
		posRadD[index] = posRad;
		return;
	}
	if (posRad.y < cMinD.y) {
		posRad.y += (cMaxD.y - cMinD.y);
		posRadD[index] = posRad;
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along z
__global__ void ApplyPeriodicBoundaryZKernel(real3 * posRadD, real4 * rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= mNumSpheresD) {
		return;
	}
	real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	} //no need to do anything if it is a boundary particle
	real3 posRad = posRadD[index];
	if (posRad.z > cMaxD.z) {
		posRad.z -= (cMaxD.z - cMinD.z);
		posRadD[index] = posRad;
		return;
	}
	if (posRad.z < cMinD.z) {
		posRad.z += (cMaxD.z - cMinD.z);
		posRadD[index] = posRad;
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along x, for ridid bodies
__global__ void ApplyPeriodicBoundaryXKernel_RigidBodies(real3 * posRigidD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numRigidBodiesD) {
		return;
	}
	real3 posRigid = posRigidD[index];
	if (posRigid.x > cMaxD.x) {
		posRigid.x -= (cMaxD.x - cMinD.x);
		posRigidD[index] = posRigid;
		return;
	}
	if (posRigid.x < cMinD.x) {
		posRigid.x += (cMaxD.x - cMinD.x);
		posRigidD[index] = posRigid;
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along x, for ridid bodies
__global__ void ApplyPeriodicBoundaryYKernel_RigidBodies(real3 * posRigidD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numRigidBodiesD) {
		return;
	}
	real3 posRigid = posRigidD[index];
	if (posRigid.y > cMaxD.y) {
		posRigid.y -= (cMaxD.y - cMinD.y);
		posRigidD[index] = posRigid;
		return;
	}
	if (posRigid.y < cMinD.y) {
		posRigid.y += (cMaxD.y - cMinD.y);
		posRigidD[index] = posRigid;
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//applies periodic BC along x, for ridid bodies
__global__ void ApplyPeriodicBoundaryZKernel_RigidBodies(real3 * posRigidD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numRigidBodiesD) {
		return;
	}
	real3 posRigid = posRigidD[index];
	if (posRigid.z > cMaxD.z) {
		posRigid.z -= (cMaxD.z - cMinD.z);
		posRigidD[index] = posRigid;
		return;
	}
	if (posRigid.z < cMinD.z) {
		posRigid.z += (cMaxD.z - cMinD.z);
		posRigidD[index] = posRigid;
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
//related to post processing of Segre-Silberberg. Distribution thing!
__global__ void PassesFromTheEnd_Kernel(
		real3 * posRigidD,
		uint * radialPositions,
		uint * radialPosCounter,
		real2 pipeCenter,
		real_ dR) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numRigidBodiesD) {
		return;
	}
	real3 posRigid = posRigidD[index];
	if ( (posRigid.x > cMaxD.x) || (posRigid.x < cMinD.x) ) {													//assuming the fluid flows in the positive x direction
		real_ r = length(R2(posRigid.y, posRigid.z) - pipeCenter);
		uint radPosition = int(r / dR);
		radialPositions[index] = radPosition;
		radialPosCounter[index] = 1;
			//printf("passed. r %f  dR %f    r/dR %f    radial_pos: %d",  r, dR , r/dR, radPosition);
		return;
	}
	//syncthreads();
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void AddToCumulutaiveNumberOfPasses(
		int * distributionD,
		uint * dummy_radialPosition,
		uint * radialPosCounter_Cumulative,
		int numberOfSections) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numberOfSections) {
		return;
	}
	uint radPosition = dummy_radialPosition[index];
	uint distributionCumul = radialPosCounter_Cumulative[index];
	if (radPosition < numberOfSections) {
		//if (distributionCumul > 0) printf("radPositon %d\n", radPosition);
		distributionD[radPosition] += distributionCumul;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void SumSurfaceInteractionForces(real3 * totalForcesRigid3, real4 * totalSurfaceInteractionRigid4, real4 * velMassRigidD) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numRigidBodiesD) {
		return;
	}
	real4 dummyVelMas = velMassRigidD[rigidSphereA];
	real3 derivRigid = rigid_SPH_massD / dummyVelMas.w * R3(totalSurfaceInteractionRigid4[rigidSphereA]);
	//** tweak 3
	if (length(derivRigid) > .2 * HSML / (dTD * dTD)) {
			derivRigid *= ( .2 * HSML / (dTD * dTD) ) / length(derivRigid);
	}
	//** end tweak
	totalForcesRigid3[rigidSphereA] = derivRigid;
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void CalcTorqueShare(real3* torqueParticlesD, real4* derivVelRhoD, real3* posRadD, int* rigidIdentifierD, real3* posRigidD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint rigidMarkerIndex = index + startRigidMarkersD;
	if (index >= numRigid_SphMarkersD) {
		return;
	}
	real3 dist3 = Distance(posRadD[rigidMarkerIndex], posRigidD[rigidIdentifierD[index]]);
	torqueParticlesD[index] = cross(dist3, R3(derivVelRhoD[rigidMarkerIndex]));
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void MapForcesOnNodes(
		real3* flexNodesForcesAllMarkers1,
		real3* flexNodesForcesAllMarkers2,
		int* flexIdentifierD,
//		int* ANCF_NumNodes_Per_Beam,
		int* ANCF_NumMarkers_Per_Beam,
		int* ANCF_NumMarkers_Per_Beam_cumul, //exclusive scan
//		int* ANCF_NumNodesMultMarkers_Per_Beam,
		int* ANCF_NumNodesMultMarkers_Per_Beam_Cumul, //exclusive scan
		real_* parametricDist,
		real4* derivVelRhoD,
		real_ markerMass)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numFlex_SphMarkersD) {
		return;
	}
	uint absMarkerIndex = index + startFlexMarkersD;
	real_ s = parametricDist[index];

	real3 derivVel = F3( derivVelRhoD[absMarkerIndex] );
	real3 markerForce = markerMass * derivVel;

	int flexBodyIndex = flexIdentifierD[index];
	int numFlexMarkersPreviousBeamsTotal = ANCF_NumMarkers_Per_Beam_cumul[flexBodyIndex];
	int numSavedForcesSoFar = ANCF_NumNodesMultMarkers_Per_Beam_Cumul[flexBodyIndex];
		int markerIndexOnThisBeam = index - numFlexMarkersPreviousBeamsTotal

		int numMarkersOnThisBeam = ANCF_NumMarkers_Per_Beam[flexBodyIndex];

	//TODO: Map Marker Force to ANCF Nodes, gives you as many forces as the number of nodes per beam
//	F0, F1, ..., F(m-1) : Forces on nodes 0, 1, 2, ..., m-1
//	Fi ---> flexNodesForces[numSavedForcesSoFar + (i * numMarkersOnThisBeam + markerIndexOnThisBeam)];
	//...
	///////

}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Populate_RigidSPH_MeshPos_LRF_kernel(
		real3* rigidSPH_MeshPos_LRF_D,
		real3* posRadD,
		int* rigidIdentifierD,
		real3* posRigidD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint rigidMarkerIndex = index + startRigidMarkersD; // updatePortionD = [start, end] index of the update portion
	if (index >= numRigid_SphParticlesD) {
		return;
	}
	real3 dist3 = posRadD[rigidMarkerIndex] - posRigidD[rigidIdentifierD[index]];
	rigidSPH_MeshPos_LRF_D[index] = dist3;
}
//--------------------------------------------------------------------------------------------------------------------------------

__global__ void Populate_FlexSPH_MeshPos_LRF_kernel(
		real3* flexSPH_MeshPos_LRF_D,
		real3 * posRadD,
		int* flexIdentifierD,
		real_* parametricDist,
		real_* ANCF_Beam_Length,
		int* ANCF_NumNodes_Per_Beam,
		real3 * ANCF_Nodes,
		real3 * ANCF_Slopes) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numFlex_SphMarkersD) {
		return;
	}
	uint absMarkerIndex = index + startFlexMarkersD; // updatePortionD = [start, end] index of the update portion
	real_ s = parametricDist[index];
	int flexBodyIndex = flexIdentifierD[index];
	real_ l = ANCF_Beam_Length[flexBodyIndex];
	int nNodes = ANCF_NumNodes_Per_Beam[flexBodyIndex];

	int indexOfClosestNode = int(s / l) * nNodes;
	if (indexOfClosestNode == nNodes) indexOfClosestNode--;

	real3 beamPointPos = Calc_ANCF_Point_Pos(ANCF_Nodes, ANCF_Slopes, indexOfClosestNode, s, l); //interpolation using ANCF beam, cubic hermit equation

	real3 dist3 = posRadD[absMarkerIndex] - beamPointPos;
	flexSPH_MeshPos_LRF_D[index] = dist3;
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void Populate_FlexSPH_MeshSlope_LRF_kernel(
		real3* flexSPH_MeshSlope_Initial_D,
		int* flexIdentifierD,
		real_* parametricDist,
		real_* ANCF_Beam_Length,
		int* ANCF_NumNodes_Per_Beam,
		real3 * ANCF_Nodes,
		real3 * ANCF_Slopes) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numFlex_SphMarkersD) {
		return;
	}
	uint absMarkerIndex = index + startFlexMarkersD; // updatePortionD = [start, end] index of the update portion
	real_ s = parametricDist[index];
	int flexBodyIndex = flexIdentifierD[index];
	real_ l = ANCF_Beam_Length[flexBodyIndex];
	int nNodes = ANCF_NumNodes_Per_Beam[flexBodyIndex];

	int indexOfClosestNode = int(s / l) * nNodes;
	if (indexOfClosestNode == nNodes) indexOfClosestNode--;

	real3 beamPointSlope = Calc_ANCF_Point_Slope(ANCF_Nodes, ANCF_Slopes, indexOfClosestNode, s, l); //interpolation using ANCF beam, cubic hermit equation
	flexSPH_MeshSlope_Initial_D[index] = beamPointSlope;
}

//--------------------------------------------------------------------------------------------------------------------------------
//the rigid body torque has been calculated in global RF. This kernel maps it to local RF to be appropriate for the formulas
//local torque = T' = A' * T
__global__ void MapTorqueToLRFKernel(real3 * AD1, real3 * AD2, real3 * AD3, real3 * totalTorque3, real3 * LF_totalTorque3) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numRigidBodiesD) {
		return;
	}
	real3 totalTorqueGRF = totalTorque3[rigidSphereA];
	LF_totalTorque3[rigidSphereA] = AD1[rigidSphereA] * totalTorqueGRF.x + AD2[rigidSphereA] * totalTorqueGRF.y
			+ AD3[rigidSphereA] * totalTorqueGRF.z;
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the rigid body particles
__global__ void UpdateKernelRigidTranstalation(real3 * totalForcesRigid3, real3 * posRigidD, real3 * posRigidCumulativeD, real4 * velMassRigidD) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numRigidBodiesD) {
		return;
	}

	real3 dummyPos = posRigidD[rigidSphereA];
	real4 dummyVelMas = velMassRigidD[rigidSphereA];

	real3 derivV_SPH = totalForcesRigid3[rigidSphereA]; //in fact, totalBodyForce4 is originially sum of dV/dt of sph particles and should be multiplied by m to produce force. gravity is applied in the force kernel

	real3 deltaPos = R3(dummyVelMas) * dTD;
	dummyPos += deltaPos;
	posRigidD[rigidSphereA] = dummyPos;
	posRigidCumulativeD[rigidSphereA] += deltaPos;

	real3 deltaVel = derivV_SPH * dTD;
	dummyVelMas += R4(deltaVel, 0);
	velMassRigidD[rigidSphereA] = dummyVelMas;
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the rigid body particles
__global__ void UpdateKernelRigidTranstalationBeta(real3 * totalForcesRigid3, real3 * posRigidD, real3 * posRigidCumulativeD, real4 * velMassRigidD) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numRigidBodiesD) {
		return;
	}

	real3 dummyPos = posRigidD[rigidSphereA];
	real4 dummyVelMas = velMassRigidD[rigidSphereA];

	real3 derivV_SPH = totalForcesRigid3[rigidSphereA]; //in fact, totalBodyForce4 is originially sum of dV/dt of sph particles and should be multiplied by m to produce force. gravity is applied in the force kernel
	derivV_SPH.y = 0;
	derivV_SPH.z = 0;

	real3 deltaPos = R3(dummyVelMas) * dTD;
	dummyPos += deltaPos;
	posRigidD[rigidSphereA] = dummyPos;
	posRigidCumulativeD[rigidSphereA] += deltaPos;

	real3 deltaVel = derivV_SPH * dTD;
	dummyVelMas += R4(deltaVel, 0);
	velMassRigidD[rigidSphereA] = dummyVelMas;
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the rigid body Quaternion of Rotation
// A is rotation matrix, A = [AD1; AD2; AD3]
__global__ void UpdateRigidBodyQuaternion_kernel(real4 * qD, real3 * omegaLRF_D) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numRigidBodiesD) {
		return;
	}
	real3 omega = omegaLRF_D[rigidSphereA];
	real4 q = qD[rigidSphereA];
	real4 qDot = 0.5 * (
			omega.x * R4(-(q.y), q.x, q.w, -(q.z)) + omega.y * R4(-(q.z), -(q.w), q.x, q.y) + omega.z * R4(-(q.w), q.z, -(q.y), q.x)
	);

	q += dTD * qDot;
	q *= (1.0f / length(q));
	qD[rigidSphereA] = q;
}
//--------------------------------------------------------------------------------------------------------------------------------
__device__ inline void RotationMatirixFromQuaternion_kernelD(real3 & AD1, real3 & AD2, real3 & AD3, const real4 & q) {
	AD1 = 2 * R3(0.5f - q.z * q.z - q.w * q.w, q.y * q.z - q.x * q.w, q.y * q.w + q.x * q.z);
	AD2 = 2 * R3(q.y * q.z + q.x * q.w, 0.5f - q.y * q.y - q.w * q.w, q.z * q.w - q.x * q.y);
	AD3 = 2 * R3(q.y * q.w - q.x * q.z, q.z * q.w + q.x * q.y, 0.5f - q.y * q.y - q.z * q.z);
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the rigid body Rotation
// A is rotation matrix, A = [AD1; AD2; AD3], first comp of q is rotation, last 3 components are axis of rot
// in wikipedia, last quat comp is the angle, in my version, first one is the angle.
// here is the mapping between wikipedia (g) and mine (q): [gx, gy, gz, gw] = [qy, qz, qw, qx]
__global__ void RotationMatirixFromQuaternion_kernel(real3 * AD1, real3 * AD2, real3 * AD3, real4 * qD) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numRigidBodiesD) {
		return;
	}
	real4 q = qD[rigidSphereA];
	AD1[rigidSphereA] = 2 * R3(0.5f - q.z * q.z - q.w * q.w, q.y * q.z - q.x * q.w, q.y * q.w + q.x * q.z);
	AD2[rigidSphereA] = 2 * R3(q.y * q.z + q.x * q.w, 0.5f - q.y * q.y - q.w * q.w, q.z * q.w - q.x * q.y);
	AD3[rigidSphereA] = 2 * R3(q.y * q.w - q.x * q.z, q.z * q.w + q.x * q.y, 0.5f - q.y * q.y - q.z * q.z);
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void UpdateRigidBodyAngularVelocity_kernel(
		real3 * LF_totalTorque3,
		real3 * jD1,
		real3 * jD2,
		real3 * jInvD1,
		real3 * jInvD2,
		real3 * omegaLRF_D) {
	uint rigidSphereA = blockIdx.x * blockDim.x + threadIdx.x;
	if (rigidSphereA >= numRigidBodiesD) {
		return;
	}

	real3 omega3 = omegaLRF_D[rigidSphereA];
	real3 j1 = jD1[rigidSphereA];
	real3 j2 = jD2[rigidSphereA];
	//printf("j j %f %f %f %f %f %f\n", j1.x, j1.y, j1.z, j2.x, j2.y, j2.z);
	real3 torquingTerm;
	torquingTerm.x = (-omega3.z * j1.y + omega3.y * j1.z) * omega3.x + (-omega3.z * j2.x + omega3.y * j2.y) * omega3.y
			+ (-omega3.z * j2.y + omega3.y * j2.z) * omega3.z;
	torquingTerm.y = (omega3.z * j1.x - omega3.x * j1.z) * omega3.x + (omega3.z * j1.y - omega3.x * j2.y) * omega3.y
			+ (omega3.z * j1.z - omega3.x * j2.z) * omega3.z;
	torquingTerm.z = (-omega3.y * j1.x + omega3.x * j1.y) * omega3.x + (-omega3.y * j1.y + omega3.x * j2.x) * omega3.y
			+ (-omega3.y * j1.z + omega3.x * j2.y) * omega3.z;

	torquingTerm = rigid_SPH_massD * LF_totalTorque3[rigidSphereA] - torquingTerm;
	//*** from this point j1 and j2 will represent the j_Inverse
	j1 = jInvD1[rigidSphereA];
	j2 = jInvD2[rigidSphereA];
	//printf("j j %f %f %f %f %f %f\n", j1.x, j1.y, j1.z, j2.x, j2.y, j2.z);
	real3 omegaDot3 = torquingTerm.x * j1 + torquingTerm.y * R3(j1.y, j2.x, j2.y) + torquingTerm.z * R3(j1.z, j2.y, j2.z);
//	//	*** for 2D motion
//		omegaDot3.x = 0;
//		omegaDot3.z = 0;

	omega3 += omegaDot3 * dTD;
	omegaLRF_D[rigidSphereA] = omega3;
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the rigid body particles
__global__ void UpdateRigidMarkersPosition(
		real3 * posRadD,
		real4 * velMasD,
		const real3 * rigidSPH_MeshPos_LRF_D,
		const int * rigidIdentifierD,
		real3 * posRigidD,
		real4 * velMassRigidD,
		real3 * omegaLRF_D,
		real3 * AD1,
		real3 * AD2,
		real3 * AD3) {

	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numRigid_SphMarkersD) {
		return;
	}
	uint rigidMarkerIndex = index + startRigidMarkersD; // updatePortionD = [start, end] index of the update portion
	int rigidBodyIndex = rigidIdentifierD[index];

	real3 a1, a2, a3;
	a1 = AD1[rigidBodyIndex];
	a2 = AD2[rigidBodyIndex];
	a3 = AD3[rigidBodyIndex];

	real3 rigidSPH_MeshPos_LRF = rigidSPH_MeshPos_LRF_D[index];

	//position
	real3 p_Rigid = posRigidD[rigidBodyIndex];
	posRadD[rigidMarkerIndex] = p_Rigid + R3(dot(a1, rigidSPH_MeshPos_LRF), dot(a2, rigidSPH_MeshPos_LRF), dot(a3, rigidSPH_MeshPos_LRF));

	//velociy
	real4 vM = velMasD[rigidMarkerIndex];
	real4 vM_Rigid = velMassRigidD[rigidBodyIndex];
	real3 omega3 = omegaLRF_D[rigidBodyIndex];
	real3 omegaCrossS = cross(omega3, rigidSPH_MeshPos_LRF);
	velMasD[rigidMarkerIndex] = R4(R3(vM_Rigid) + R3(dot(a1, omegaCrossS), dot(a2, omegaCrossS), dot(a3, omegaCrossS)), vM.w);
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the flex body markers
__global__ void UpdateFlexMarkersPosition(
		real3 * posRadD,
		real4 * velMasD,
		int* flexIdentifierD,
		real3* flexSPH_MeshPos_LRF_D,
		real3* flexSPH_MeshSlope_Initial_D,
		real_* parametricDist,
		real_* ANCF_Beam_Length,
		int* ANCF_NumNodes_Per_Beam,
		real3 * ANCF_Nodes,
		real3 * ANCF_Slopes,
		real3 * ANCF_VelNodes,
		real3 * ANCF_VelSlopes) {

	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numFlex_SphMarkersD) {
		return;
	}
	uint absMarkerIndex = index + startFlexMarkersD; // updatePortionD = [start, end] index of the update portion
	real_ s = parametricDist[index];
	int flexBodyIndex = flexIdentifierD[index];
	real_ l = ANCF_Beam_Length[flexBodyIndex];
	int nNodes = ANCF_NumNodes_Per_Beam[flexBodyIndex];

	int indexOfClosestNode = int(s / l) * nNodes;
	if (indexOfClosestNode == nNodes) indexOfClosestNode--;

	real3 beamPointPos = Calc_ANCF_Point_Pos(ANCF_Nodes, ANCF_Slopes, indexOfClosestNode, s, l); //interpolation using ANCF beam, cubic hermit equation
	real3 beamPointSlope = Calc_ANCF_Point_Slope(ANCF_Nodes, ANCF_Slopes, indexOfClosestNode, s, l); //interpolation using ANCF beam, cubic hermit equation
	real3 beamPointOmega;

	real3 beamPointVel = Calc_ANCF_Point_Vel(ANCF_Nodes, ANCF_Slopes, ANCF_VelNodes, ANCF_VelSlopes, indexOfClosestNode, s, l); //interpolation using ANCF beam, cubic hermit equation

	real3 dist3 = flexSPH_MeshPos_LRF_D[index];
	real3 beamPointSlopeInitial = flexSPH_MeshSlope_Initial_D[index];
	//Important Important Important Important Important Important Important Important Important
	//Important Important Important Important Important Important Important Important Important
	//Important Important Important Important Important Important Important Important Important
	// Assumed Calc_ANCF_Point_Slope returns the unit vector. theta calculation is based on this assumption. Also cross product
	real_ theta = acos(dot(beamPointSlopeInitial, beamPointSlope));
	real3 n3 = cross(beamPointSlopeInitial, beamPointSlope);
	n3 /= length(n3);
	real4 q = R4(cos(0.5 * theta),
			n3.x * sin(0.5 * theta), n3.y * sin(0.5 * theta), n3.z * sin(0.5 * theta));
	real3 A1, A2, A3;
	RotationMatirixFromQuaternion_kernelD(A1, A2, A3, q);
	posRadD[absMarkerIndex] = beamPointPos + R3(dot(A1, dist3), dot(A2, dist3), dot(A3, dist3));
	real3 absOmega = 	Calc_ANCF_Point_Omega(ANCF_Nodes, ANCF_Slopes, ANCF_VelNodes, ANCF_VelSlopes, indexOfClosestNode, s, l); //interpolation using ANCF beam, cubic hermit equation
	velMasD[absMarkerIndex] = beamPointVel + cross(absOmega, dist3);

}
//--------------------------------------------------------------------------------------------------------------------------------
void MapSPH_ToGrid(
		real_ resolution,
		int3 & cartesianGridDims,
		thrust::host_vector<real4> & rho_Pres_CartH,
		thrust::host_vector<real4> & vel_VelMag_CartH,
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real4> & rhoPresMuD,
		int mNSpheres,
		SimParams paramsH) {
//	real3* m_dSortedPosRad;
//	real4* m_dSortedVelMas;
//	real4* m_dSortedRhoPreMu;
//	uint* m_dCellStart; // index of start of each cell in sorted list
//	uint* m_dCellEnd; // index of end of cell

	int3 SIDE = paramsH.gridSize;
	uint m_numGridCells = SIDE.x * SIDE.y * SIDE.z; //m_gridSize = SIDE
	//TODO here

	// calculate grid hash
	thrust::device_vector<real3> m_dSortedPosRad(mNSpheres);
	thrust::device_vector<real4> m_dSortedVelMas(mNSpheres);
	thrust::device_vector<real4> m_dSortedRhoPreMu(mNSpheres);

	thrust::device_vector<uint> m_dGridParticleHash(mNSpheres);
	thrust::device_vector<uint> m_dGridParticleIndex(mNSpheres);

	thrust::device_vector<uint> m_dCellStart(m_numGridCells);
	thrust::device_vector<uint> m_dCellEnd(m_numGridCells);

	// calculate grid hash
	calcHash(U1CAST(m_dGridParticleHash), U1CAST(m_dGridParticleIndex), R3CAST(posRadD), mNSpheres);

	thrust::sort_by_key(m_dGridParticleHash.begin(), m_dGridParticleHash.end(), m_dGridParticleIndex.begin());

	// reorder particle arrays into sorted order and find start and end of each cell
	reorderDataAndFindCellStart(U1CAST(m_dCellStart), U1CAST(m_dCellEnd), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu), U1CAST(m_dGridParticleHash),
			U1CAST(m_dGridParticleIndex), TCAST(posRadD), R4CAST(velMasD), R4CAST(rhoPresMuD), mNSpheres, m_numGridCells);

	//real_ resolution = 8 * paramsH.particleRadius;
	cartesianGridDims = I3(paramsH.boxDims / resolution) + I3(1);
//	printf("^^^ bodDim %f %f %f, GridDim %d %d %d, resolution %f \n", paramsH.boxDims.x, paramsH.boxDims.y, paramsH.boxDims.z, cartesianGridDims.x,
//			cartesianGridDims.y, cartesianGridDims.z, resolution);
	uint cartesianGridSize = cartesianGridDims.x * cartesianGridDims.y * cartesianGridDims.z;
	thrust::device_vector<real4> rho_Pres_CartD(cartesianGridSize);
	thrust::device_vector<real4> vel_VelMag_CartD(cartesianGridSize);

	CalcCartesianData(R4CAST(rho_Pres_CartD), R4CAST(vel_VelMag_CartD), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu),
			U1CAST(m_dGridParticleIndex), U1CAST(m_dCellStart), U1CAST(m_dCellEnd), cartesianGridSize, cartesianGridDims, resolution);

//	freeArray(m_dSortedPosRad);
//	freeArray(m_dSortedVelMas);
//	freeArray(m_dSortedRhoPreMu);
	m_dSortedPosRad.clear();
	m_dSortedVelMas.clear();
	m_dSortedRhoPreMu.clear();

	m_dGridParticleHash.clear();
	m_dGridParticleIndex.clear();

//	freeArray(m_dCellStart);
//	freeArray(m_dCellEnd);
	m_dCellStart.clear();
	m_dCellEnd.clear();

	rho_Pres_CartH.resize(cartesianGridSize);
	vel_VelMag_CartH.resize(cartesianGridSize);
	thrust::copy(rho_Pres_CartD.begin(), rho_Pres_CartD.end(), rho_Pres_CartH.begin());
	thrust::copy(vel_VelMag_CartD.begin(), vel_VelMag_CartD.end(), vel_VelMag_CartH.begin());

	rho_Pres_CartD.clear();
	vel_VelMag_CartD.clear();
}

//*******************************************************************************************************************************
//builds the neighbors' list of each particle and finds the force on each particle
//calculates the interaction force between 1- fluid-fluid, 2- fluid-solid, 3- solid-fluid particles
//calculates forces from other SPH or solid particles, as wall as boundaries
void ForceSPH(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real3> & vel_XSPH_D,
		thrust::device_vector<real4> & rhoPresMuD,
		thrust::device_vector<uint> & bodyIndexD,
		thrust::device_vector<real4> & derivVelRhoD,
		const thrust::host_vector<int2> & referenceArray,
		int mNSpheres,
		int3 SIDE,
		real_ dT) {
	// Part1: contact detection #########################################################################################################################
	// grid data for sorting method
//	real3* m_dSortedPosRad;
//	real4* m_dSortedVelMas;
//	real4* m_dSortedRhoPreMu;
//	uint* m_dCellStart; // index of start of each cell in sorted list
//	uint* m_dCellEnd; // index of end of cell

	uint m_numGridCells = SIDE.x * SIDE.y * SIDE.z; //m_gridSize = SIDE
	//TODO here

	// calculate grid hash
	thrust::device_vector<real3> m_dSortedPosRad(mNSpheres);
	thrust::device_vector<real4> m_dSortedVelMas(mNSpheres);
	thrust::device_vector<real4> m_dSortedRhoPreMu(mNSpheres);
	thrust::device_vector<real3> vel_XSPH_Sorted_D(mNSpheres);

	thrust::device_vector<uint> m_dGridParticleHash(mNSpheres);
	thrust::device_vector<uint> m_dGridParticleIndex(mNSpheres);

	thrust::device_vector<uint> m_dCellStart(m_numGridCells);
	thrust::device_vector<uint> m_dCellEnd(m_numGridCells);
	// calculate grid hash
	calcHash(U1CAST(m_dGridParticleHash), U1CAST(m_dGridParticleIndex), R3CAST(posRadD), mNSpheres);

	thrust::sort_by_key(m_dGridParticleHash.begin(), m_dGridParticleHash.end(), m_dGridParticleIndex.begin());

	// reorder particle arrays into sorted order and find start and end of each cell
	reorderDataAndFindCellStart(U1CAST(m_dCellStart), U1CAST(m_dCellEnd), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu), U1CAST(m_dGridParticleHash),
			U1CAST(m_dGridParticleIndex), TCAST(posRadD), R4CAST(velMasD), R4CAST(rhoPresMuD), mNSpheres, m_numGridCells);

	//process collisions
	real4 totalFluidBodyForce4 = bodyForce4 + R4(Gravity);
	thrust::fill(derivVelRhoD.begin(), derivVelRhoD.end(), R4(0)); //initialize derivVelRhoD with zero. necessary
	thrust::fill(derivVelRhoD.begin() + referenceArray[0].x, derivVelRhoD.begin() + referenceArray[0].y, totalFluidBodyForce4); //add body force to fluid particles.

	RecalcVelocity_XSPH(R3CAST(vel_XSPH_Sorted_D), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu), U1CAST(m_dGridParticleIndex), U1CAST(m_dCellStart),
			U1CAST(m_dCellEnd), mNSpheres, m_numGridCells);

	collide(R4CAST(derivVelRhoD), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R3CAST(vel_XSPH_Sorted_D), R4CAST(m_dSortedRhoPreMu), U1CAST(m_dGridParticleIndex), U1CAST(m_dCellStart),
			U1CAST(m_dCellEnd), mNSpheres, m_numGridCells, dT);


	uint nBlock_NumSpheres, nThreads_SphMarkers;
	computeGridSize(mNSpheres, 256, nBlock_NumSpheres, nThreads_SphMarkers);
	Copy_SortedVelXSPH_To_VelXSPH<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(R3CAST(vel_XSPH_D), R3CAST(vel_XSPH_Sorted_D), U1CAST(m_dGridParticleIndex), mNSpheres);
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: Copy_SortedVelXSPH_To_VelXSPH");

	////
	m_dSortedPosRad.clear();
	m_dSortedVelMas.clear();
	m_dSortedRhoPreMu.clear();
	vel_XSPH_Sorted_D.clear();

	m_dGridParticleHash.clear();
	m_dGridParticleIndex.clear();

	m_dCellStart.clear();
	m_dCellEnd.clear();
}
//--------------------------------------------------------------------------------------------------------------------------------
void DensityReinitialization(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real4> & rhoPresMuD,
		int mNSpheres,
		int3 SIDE) {
//	real3* m_dSortedPosRad;
//	real4* m_dSortedVelMas;
//	real4* m_dSortedRhoPreMu;
//	uint* m_dCellStart; // index of start of each cell in sorted list
//	uint* m_dCellEnd; // index of end of cell

	uint m_numGridCells = SIDE.x * SIDE.y * SIDE.z; //m_gridSize = SIDE
	//TODO here

	// calculate grid hash
	thrust::device_vector<real3> m_dSortedPosRad(mNSpheres);
	thrust::device_vector<real4> m_dSortedVelMas(mNSpheres);
	thrust::device_vector<real4> m_dSortedRhoPreMu(mNSpheres);

	thrust::device_vector<uint> m_dGridParticleHash(mNSpheres);
	thrust::device_vector<uint> m_dGridParticleIndex(mNSpheres);

	thrust::device_vector<uint> m_dCellStart(m_numGridCells);
	thrust::device_vector<uint> m_dCellEnd(m_numGridCells);

	// calculate grid hash
	calcHash(U1CAST(m_dGridParticleHash), U1CAST(m_dGridParticleIndex), R3CAST(posRadD), mNSpheres);

	thrust::sort_by_key(m_dGridParticleHash.begin(), m_dGridParticleHash.end(), m_dGridParticleIndex.begin());

	// reorder particle arrays into sorted order and find start and end of each cell
	reorderDataAndFindCellStart(U1CAST(m_dCellStart), U1CAST(m_dCellEnd), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu), U1CAST(m_dGridParticleHash),
			U1CAST(m_dGridParticleIndex), TCAST(posRadD), R4CAST(velMasD), R4CAST(rhoPresMuD), mNSpheres, m_numGridCells);

	ReCalcDensity(R3CAST(posRadD), R4CAST(velMasD), R4CAST(rhoPresMuD), R3CAST(m_dSortedPosRad), R4CAST(m_dSortedVelMas), R4CAST(m_dSortedRhoPreMu),
			U1CAST(m_dGridParticleIndex), U1CAST(m_dCellStart), U1CAST(m_dCellEnd), mNSpheres, m_numGridCells);

	m_dSortedPosRad.clear();
	m_dSortedVelMas.clear();
	m_dSortedRhoPreMu.clear();

	m_dGridParticleHash.clear();
	m_dGridParticleIndex.clear();

	m_dCellStart.clear();
	m_dCellEnd.clear();
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the fluid particles by calling UpdateKernelFluid 
void UpdateFluid(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real3> & vel_XSPH_D,
		thrust::device_vector<real4> & rhoPresMuD,
		thrust::device_vector<real4> & derivVelRhoD,
		const thrust::host_vector<int2> & referenceArray,
		real_ dT) {
	int2 updatePortion = referenceArray[0];
	//int2 updatePortion = I2(referenceArray[0].x, referenceArray[0].y);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dTD), &dT, sizeof(dT));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(updatePortionD), &updatePortion, sizeof(updatePortion));

	uint nBlock_UpdateFluid, nThreads;
	computeGridSize(updatePortion.y - updatePortion.x, 128, nBlock_UpdateFluid, nThreads);
	UpdateKernelFluid<<<nBlock_UpdateFluid, nThreads>>>(R3CAST(posRadD), R4CAST(velMasD), R3CAST(vel_XSPH_D), R4CAST(rhoPresMuD), R4CAST(derivVelRhoD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelFluid");
}
//--------------------------------------------------------------------------------------------------------------------------------
//updates the fluid particles by calling UpdateBoundary
void UpdateBoundary(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real4> & rhoPresMuD,
		thrust::device_vector<real4> & derivVelRhoD,
		const thrust::host_vector<int2> & referenceArray,
		real_ dT) {
	int2 updatePortion = referenceArray[1];
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dTD), &dT, sizeof(dT));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(updatePortionD), &updatePortion, sizeof(updatePortion));

	uint nBlock_UpdateFluid, nThreads;
	computeGridSize(updatePortion.y - updatePortion.x, 128, nBlock_UpdateFluid, nThreads);
	UpdateKernelBoundary<<<nBlock_UpdateFluid, nThreads>>>(R3CAST(posRadD), R4CAST(velMasD), R4CAST(rhoPresMuD), R4CAST(derivVelRhoD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelBoundary");
}
//--------------------------------------------------------------------------------------------------------------------------------
void ApplyBoundary(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & rhoPresMuD,
		int mNSpheres,
		thrust::device_vector<real3> & posRigidD,
		thrust::device_vector<real4> & velMassRigidD,
		int numRigidBodies) {
	uint nBlock_NumSpheres, nThreads_SphMarkers;
	computeGridSize(mNSpheres, 256, nBlock_NumSpheres, nThreads_SphMarkers);
	ApplyPeriodicBoundaryXKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(R3CAST(posRadD), R4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: ApplyPeriodicBoundaryXKernel");
	// these are useful anyway for out of bound particles
	ApplyPeriodicBoundaryYKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(R3CAST(posRadD), R4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: ApplyPeriodicBoundaryXKernel");
	ApplyPeriodicBoundaryZKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(R3CAST(posRadD), R4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: ApplyPeriodicBoundaryXKernel");
//////////////
	uint nBlock_NumRigids, nThreads_RigidBodies;
	computeGridSize(numRigidBodies, 128, nBlock_NumRigids, nThreads_RigidBodies);

	hipMemcpyToSymbolAsync(HIP_SYMBOL(numRigidBodiesD), &numRigidBodies, sizeof(numRigidBodies)); //can be defined outside of the kernel, and only once
	ApplyPeriodicBoundaryXKernel_RigidBodies<<<nBlock_NumRigids, nThreads_RigidBodies>>>(R3CAST(posRigidD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");
	ApplyPeriodicBoundaryYKernel_RigidBodies<<<nBlock_NumRigids, nThreads_RigidBodies>>>(R3CAST(posRigidD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");
	ApplyPeriodicBoundaryZKernel_RigidBodies<<<nBlock_NumRigids, nThreads_RigidBodies>>>(R3CAST(posRigidD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");
}
//--------------------------------------------------------------------------------------------------------------------------------
void FindPassesFromTheEnd(
		thrust::device_vector<real3> & posRigidD,
		thrust::device_vector<int> & distributionD,
		int numRigidBodies,
		real2 pipeCenter,
		real_ pipeRadius,
		int numberOfSections) {
//	real3 posRigid = posRigidD[0];
//	printf("xRigid %f\n", posRadRigid.x);cutil_math deprecate
	real_ dR = pipeRadius / numberOfSections;
	thrust::device_vector<uint> radialPositions(numRigidBodies);
	thrust::device_vector<uint> radialPosCounter(numRigidBodies);
	thrust::fill(radialPositions.begin(), radialPositions.end(), 10000); //10000 as a large number
	thrust::fill(radialPosCounter.begin(), radialPosCounter.end(), 0);

	uint nBlock_NumRigids, nThreads_RigidBodies;
	computeGridSize(numRigidBodies, 128, nBlock_NumRigids, nThreads_RigidBodies);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(numRigidBodiesD), &numRigidBodies, sizeof(numRigidBodies)); //can be defined outside of the kernel, and only once
	PassesFromTheEnd_Kernel<<<nBlock_NumRigids, nThreads_RigidBodies>>>(R3CAST(posRigidD), U1CAST(radialPositions), U1CAST(radialPosCounter), pipeCenter, dR);
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: PassesFromTheEnd_Kernel");

	thrust::sort_by_key(radialPositions.begin(), radialPositions.end(), radialPosCounter.begin());
	thrust::device_vector<uint> radialPosCounter_Cumulative(numberOfSections + 2); //+2 for safety, specially when the particle goes outside of the pipe
	thrust::device_vector<uint> dummy_radialPosition(numberOfSections + 2);
	(void) thrust::reduce_by_key(radialPositions.begin(), radialPositions.end(), radialPosCounter.begin(), dummy_radialPosition.begin(),
			radialPosCounter_Cumulative.begin());
//	radialPosCounter_Cumulative.resize(numberOfSections);
//	dummy_radialPosition.resize(numberOfSections);

	//printf("%$%$%$%$%$%$ dummy_radialPosition[0] %d")

	uint nBlock_NumSections, nThreads_numSections;
	computeGridSize(numberOfSections, 128, nBlock_NumSections, nThreads_numSections);
	AddToCumulutaiveNumberOfPasses<<<nBlock_NumSections, nThreads_numSections>>>(I1CAST(distributionD), U1CAST(dummy_radialPosition), U1CAST(radialPosCounter_Cumulative), numberOfSections);
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: AddToCumulutaiveNumberOfPasses");

	radialPosCounter_Cumulative.clear();
	dummy_radialPosition.clear();
	radialPositions.clear();
	radialPosCounter.clear();
}
//--------------------------------------------------------------------------------------------------------------------------------
void UpdateRigidBody(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real3> & posRigidD,
		thrust::device_vector<real3> & posRigidCumulativeD,
		thrust::device_vector<real4> & velMassRigidD,
		thrust::device_vector<real4> & qD,
		thrust::device_vector<real3> & AD1,
		thrust::device_vector<real3> & AD2,
		thrust::device_vector<real3> & AD3,
		thrust::device_vector<real3> & omegaLRF_D,
		thrust::device_vector<real4> & derivVelRhoD,
		const thrust::device_vector<int> & rigidIdentifierD,
		const thrust::device_vector<real3> & rigidSPH_MeshPos_LRF_D,
		const thrust::host_vector<int2> & referenceArray,
		const thrust::device_vector<real3> & jD1,
		const thrust::device_vector<real3> & jD2,
		const thrust::device_vector<real3> & jInvD1,
		const thrust::device_vector<real3> & jInvD2,
		SimParams paramsH,
		int numRigidBodies,
		int startRigidMarkers,
		int numRigid_SphMarkers,
		float fracSimulation,
		real_ dT) {
	if (referenceArray.size() < 3) {
		return;
	}
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dTD), &dT, sizeof(dT));
//g
	thrust::device_vector<real4> totalSurfaceInteractionRigid4(numRigidBodies);
	thrust::device_vector<real3> totalTorque3(numRigidBodies);
	thrust::fill(totalSurfaceInteractionRigid4.begin(), totalSurfaceInteractionRigid4.end(), R4(0));
	thrust::device_vector<int> dummyIdentify(numRigidBodies);
	thrust::equal_to<int> binary_pred;

	(void) thrust::reduce_by_key(rigidIdentifierD.begin(), rigidIdentifierD.end(), derivVelRhoD.begin() + startRigidMarkers, dummyIdentify.begin(),
			totalSurfaceInteractionRigid4.begin(), binary_pred, thrust::plus<real4>());

	uint nBlocks_numRigid_SphMarkers;
	uint nThreads_SphMarkers;
	computeGridSize(numRigid_SphMarkers, 256, nBlocks_numRigid_SphMarkers, nThreads_SphMarkers);

	thrust::device_vector<real3> totalForcesRigid3(numRigidBodies);
	thrust::fill(totalForcesRigid3.begin(), totalForcesRigid3.end(), R3(0));
	SumSurfaceInteractionForces<<<nBlocks_numRigid_SphMarkers, nThreads_SphMarkers>>>(R3CAST(totalForcesRigid3), R4CAST(totalSurfaceInteractionRigid4), R4CAST(velMassRigidD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: SumSurfaceInteractionForces");
	totalSurfaceInteractionRigid4.clear();



	thrust::device_vector<real3> torqueParticlesD(numRigid_SphMarkers);
	CalcTorqueShare<<<nBlocks_numRigid_SphMarkers, nThreads_SphMarkers>>>(R3CAST(torqueParticlesD), R4CAST(derivVelRhoD), R3CAST(posRadD), I1CAST(rigidIdentifierD), R3CAST(posRigidD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: CalcTorqueShare");
	(void) thrust::reduce_by_key(rigidIdentifierD.begin(), rigidIdentifierD.end(), torqueParticlesD.begin(), dummyIdentify.begin(),
			totalTorque3.begin(), binary_pred, thrust::plus<real3>());

	torqueParticlesD.clear();
	dummyIdentify.clear();

	//add gravity
	thrust::device_vector<real3> gravityForces3(numRigidBodies);
	thrust::fill(gravityForces3.begin(), gravityForces3.end(), paramsH.gravity);
	thrust::transform(totalForcesRigid3.begin(), totalForcesRigid3.end(), gravityForces3.begin(), totalForcesRigid3.begin(), thrust::plus<real3>());
	gravityForces3.clear();

	//################################################### update rigid body things
	uint nBlock_UpdateRigid;
	uint nThreads_rigidParticles;
	computeGridSize(numRigidBodies, 128, nBlock_UpdateRigid, nThreads_rigidParticles);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(numRigidBodiesD), &numRigidBodies, sizeof(numRigidBodies)); //can be defined outside of the kernel, and only once

	// copy rigid_SPH_mass to symbol -constant memory
	thrust::device_vector<real3> LF_totalTorque3(numRigidBodies);
	MapTorqueToLRFKernel<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R3CAST(AD1), R3CAST(AD2), R3CAST(AD3), R3CAST(totalTorque3), R3CAST(LF_totalTorque3));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: MapTorqueToLRFKernel");
	totalTorque3.clear();

	if (fracSimulation <.01) {
		UpdateKernelRigidTranstalationBeta<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R3CAST(totalForcesRigid3), R3CAST(posRigidD), R3CAST(posRigidCumulativeD), R4CAST(velMassRigidD));
	} else {
		UpdateKernelRigidTranstalation<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R3CAST(totalForcesRigid3), R3CAST(posRigidD), R3CAST(posRigidCumulativeD), R4CAST(velMassRigidD));
	}
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");
	totalForcesRigid3.clear();

	UpdateRigidBodyQuaternion_kernel<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R4CAST(qD), R3CAST(omegaLRF_D));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateRotation");

	RotationMatirixFromQuaternion_kernel<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R3CAST(AD1), R3CAST(AD2), R3CAST(AD3), R4CAST(qD));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateRotation");

	UpdateRigidBodyAngularVelocity_kernel<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R3CAST(LF_totalTorque3), R3CAST(jD1), R3CAST(jD2), R3CAST(jInvD1), R3CAST(jInvD2), R3CAST(omegaLRF_D));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");

	LF_totalTorque3.clear();
	//################################################### update rigid body things
	UpdateRigidMarkersPosition<<<nBlocks_numRigid_SphMarkers, nThreads_SphMarkers>>>(R3CAST(posRadD), R4CAST(velMasD), R3CAST(rigidSPH_MeshPos_LRF_D), I1CAST(rigidIdentifierD), R3CAST(posRigidD), R4CAST(velMassRigidD), R3CAST(omegaLRF_D), R3CAST(AD1), R3CAST(AD2), R3CAST(AD3));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");
}
//--------------------------------------------------------------------------------------------------------------------------------
void UpdateFlexibleBody(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		const int numRigidBodies,
		const int numFlexBodies,
		const int numFlex_SphMarkers,
		thrust::device_vector<real3> & ANCF_Nodes,
		thrust::device_vector<real3> & ANCF_Slopes,
		thrust::device_vector<real3> & ANCF_VelNodes,
		thrust::device_vector<real3> & ANCF_VelSlopes,
		thrust::device_vector<int2> & ANCF_ReferenceArrayNodesOnBeams,
		thrust::device_vector<int> & ANCF_NumNodes_Per_Beam,
		thrust::device_vector<int> & ANCF_NumMarkers_Per_Beam,
		thrust::device_vector<int> & ANCF_NumNodesMultMarkers_Per_Beam,
		thrust::device_vector<int> & ANCF_NumNodesMultMarkers_Per_Beam_Cumul,

		const thrust::device_vector<int> & flexIdentifierD,
		const thrust::device_vector<real3> & flexSPH_MeshPos_LRF_D,
		const thrust::device_vector<real_> & parametricDist,
		const thrust::device_vector<real_> & ANCF_Beam_Length,
		const thrust::host_vector<int2> & referenceArray,

		SimParams paramsH,
		float fracSimulation,
		real_ dT) {
	if (numFlexBodies == 0) {
		return;
	}
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dTD), &dT, sizeof(dT));

	int numFlBcRigid = 2 + numRigidBodies;
	int totalNumberOfFlexNodes = ANCF_ReferenceArrayNodesOnBeams[ANCF_ReferenceArrayNodesOnBeams.size() - 1].y;

	thrust::device_vector<real3> flexNodesForces1();//(totalNumberOfFlexNodes); Size:sum(numNodesOfEachBeam*numSPH_MarkersOfEachBeam)
	thrust::device_vector<real3> flexNodesForces2();

	thrust::device_vector<real3> flexNodesForcesAllMarkers1();//(totalNumberOfFlexNodes * totalNumberOfFlexMarkers); Size:sum(numNodesOfEachBeam*numSPH_MarkersOfEachBeam)
	thrust::device_vector<real3> flexNodesForcesAllMarkers2();//(totalNumberOfFlexNodes * totalNumberOfFlexMarkers);

	uint nBlocks_numFlex_SphMarkers;
	uint nThreads_SphMarkers;
	computeGridSize(numFlex_SphMarkers, 256, nBlocks_numFlex_SphMarkers, nThreads_SphMarkers);
	MapForcesOnNodes<<<nBlocks_numFlex_SphMarkers, nThreads_SphMarkers>>>(
			R3CAST(flexNodesForcesAllMarkers1),
			R3CAST(flexNodesForcesAllMarkers2),
			I1CAST(flexIdentifierD),
			I1CAST(ANCF_NumMarkers_Per_Beam),
			I1CAST(ANCF_NumMarkers_Per_Beam_cumul),
			I1CAST(ANCF_NumNodesMultMarkers_Per_Beam_Cumul),
			R1CAST(parametricDist),
			R4CAST(derivVelRhoD),
			markerMass);

	if (nodesAndFlexPairIdentifier.size() != flexNodesForcesAllMarkers1.size()) {
		printf("we have size inconsistency between flex nodesForces and nodesPair identifier");
	}
	thrust::device_vector<int> dummyNodesFlexIdentify(nodesAndFlexPairIdentifier.size());
	thrust::equal_to<int2> binary_pred_int2; //if binary_pred int2 does not work, you have to either add operator == to custom_cutil_math, or you have to map nodes identifiers from int2 to int
	(void) thrust::reduce_by_key(nodesAndFlexPairIdentifier.begin(), nodesAndFlexPairIdentifier.end(), flexNodesForcesAllMarkers1.begin(), dummyNodesFlexIdentify.begin(),
			flexNodesForces1.begin(), binary_pred_int2, thrust::plus<real3>());
	(void) thrust::reduce_by_key(nodesAndFlexPairIdentifier.begin(), nodesAndFlexPairIdentifier.end(), flexNodesForcesAllMarkers2.begin(), dummyNodesFlexIdentify.begin(),
			flexNodesForces2.begin(), binary_pred_int2, thrust::plus<real3>());
	flexNodesForcesAllMarkers1.clear();
	flexNodesForcesAllMarkers2.clear();

//	//TODO: update flex bodies here
//	 ....
//	 ....
//	 ....
//	 ....
//	//end

//	//TODO: add gravity to Flex objects
//	thrust::device_vector<real3> gravityForces3(numRigidBodies);
//	thrust::fill(gravityForces3.begin(), gravityForces3.end(), paramsH.gravity);
//	thrust::transform(totalForcesRigid3.begin(), totalForcesRigid3.end(), gravityForces3.begin(), totalForcesRigid3.begin(), thrust::plus<real3>());
//	gravityForces3.clear();
//	//

	//################################################### update rigid body things
	computeGridSize(numFlex_SphMarkers, 256, nBlocks_numFlex_SphMarkers, nThreads_SphMarkers);

	UpdateFlexMarkersPosition<<<nBlocks_numFlex_SphMarkers, nThreads_SphMarkers>>>(
			R3CAST(posRadD), R4CAST(velMasD),
			I1CAST(flexIdentifierD),
			R3CAST(flexSPH_MeshPos_LRF_D),
			R1CAST(parametricDist),
			R1CAST(ANCF_Beam_Length),
			I1CAST(ANCF_NumNodes_Per_Beam),
			R3CAST(ANCF_Nodes),
			R3CAST(ANCF_Slopes),
			R3CAST(ANCF_VelNodes),
			R3CAST(ANCF_VelSlopes)
			);

	hipDeviceSynchronize();

	CUT_CHECK_ERROR("Kernel execution failed: UpdateKernelRigid");


	//------------------------ delete stuff
	dummyNodesFlexIdentify.clear();


	flexNodesForces1.clear();
	flexNodesForces2.clear();
}
////--------------------------------------------------------------------------------------------------------------------------------
//##############################################################################################################################################
// the main function, which updates the particles and implements BC
void cudaCollisions(
		thrust::host_vector<real3> & mPosRad,
		thrust::host_vector<real4> & mVelMas,
		thrust::host_vector<real4> & mRhoPresMu,
		const thrust::host_vector<uint> & bodyIndex,
		const thrust::host_vector<int2> & referenceArray,
		const thrust::host_vector<int2> & flexIdentifier,
		int & mNSpheres,
		real3 cMax,
		real3 cMin,
		real_ delT,
		thrust::host_vector<real3> & posRigidH,
		thrust::host_vector<real4> & mQuatRot,
		thrust::host_vector<real4> & velMassRigidH,
		thrust::host_vector<real3> omegaLRF_H,
		thrust::host_vector<real3> jH1,
		thrust::host_vector<real3> jH2,
		thrust::host_vector<real3> jInvH1,
		thrust::host_vector<real3> jInvH2,
		real_ binSize0,
		real_ channelRadius,
		real2 channelCenterYZ) {
	//--------- initialization ---------------
	//hipError_t dumDevErr = hipSetDevice(2);
	GpuTimer myTotalTime;
	myTotalTime.Start();
	printf("a1 yoho\n");
	//printf("cMin.x, y, z, CMAx.x, y, z, binSize %f %f %f , %f %f %f, %f\n", cMin.x, cMin.y, cMin.z, cMax.x, cMax.y, cMax.z, binSize0); 
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	hipMemcpyToSymbolAsync(HIP_SYMBOL(cMinD), &cMin, sizeof(cMin));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(cMaxD), &cMax, sizeof(cMax));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(mNumSpheresD), &mNSpheres, sizeof(mNSpheres));
	printf("a2 yoho\n");

	int numRigidBodies = posRigidH.size();
	thrust::device_vector<real3> posRadD=mPosRad;
	//thrust::copy(mPosRad.begin(), mPosRad.end(), posRadD.begin());
	thrust::device_vector<real4> velMasD=mVelMas;
	//thrust::copy(mVelMas.begin(), mVelMas.end(), velMasD.begin());
	thrust::device_vector<real4> rhoPresMuD=mRhoPresMu;
	//thrust::copy(mRhoPresMu.begin(), mRhoPresMu.end(), rhoPresMuD.begin());
	printf("a3 yoho\n");

	thrust::device_vector<real3> posRigidD=posRigidH;
	//thrust::copy(posRigidH.begin(), posRigidH.end(), posRigidD.begin());
	thrust::device_vector<real3> posRigidCumulativeD=posRigidH;
	//thrust::copy(posRigidH.begin(), posRigidH.end(), posRigidCumulativeD.begin());
	thrust::device_vector<real4> velMassRigidD=velMassRigidH;
	//thrust::copy(velMassRigidH.begin(), velMassRigidH.end(), velMassRigidD.begin());
	thrust::device_vector<real3> omegaLRF_D=omegaLRF_H;
	//thrust::copy(omegaLRF_H.begin(), omegaLRF_H.end(), omegaLRF_D.begin());
	printf("a4 yoho\n");
	thrust::device_vector<real3> jD1=jH1;
	thrust::device_vector<real3> jD2=jH2;
	thrust::device_vector<real3> jInvD1=jInvH1;
	thrust::device_vector<real3> jInvD2=jInvH2;
	//thrust::copy(jH1.begin(), jH1.end(), jD1.begin());
	//thrust::copy(jH2.begin(), jH2.end(), jD2.begin());
	//thrust::copy(jInvH1.begin(), jInvH1.end(), jInvD1.begin());
	//thrust::copy(jInvH2.begin(), jInvH2.end(), jInvD2.begin());
	printf("a5 yoho\n");
	thrust::device_vector<uint> bodyIndexD=bodyIndex;
	//thrust::copy(bodyIndex.begin(), bodyIndex.end(), bodyIndexD.begin());
	thrust::device_vector<real4> derivVelRhoD(mNSpheres);
	printf("a6 yoho\n");
		//******************** rigid body some initialization
	thrust::device_vector<int> rigidIdentifierD(0);

	real_ rigid_SPH_mass;																					//____________________________> typical mass, save to constant memory
	int numRigid_SphMarkers = 0;
	int startRigidMarkers = (referenceArray[1]).y;
	if (referenceArray.size() > 2) {
		startRigidMarkers = (referenceArray[2]).x;
		numRigid_SphMarkers = referenceArray[2 + numRigidBodies - 1].y - startRigidMarkers;
		rigidIdentifierD.resize(numRigid_SphMarkers);
		for (int rigidSphereA = 0; rigidSphereA < numRigidBodies; rigidSphereA++) {
			int2 updatePortion = referenceArray[2 + rigidSphereA]; //first two component of the referenceArray denote to the fluid and boundary particles
			thrust::fill(rigidIdentifierD.begin() + (updatePortion.x - startRigidMarkers),
					rigidIdentifierD.begin() + (updatePortion.y - startRigidMarkers), rigidSphereA);
		}

		//---
		real4 typicalRigidSPH = mVelMas[referenceArray[2].x];
		rigid_SPH_mass = typicalRigidSPH.w;
	} else {
		real4 dummyFluid = mVelMas[referenceArray[0].x];
		rigid_SPH_mass = 100 * dummyFluid.w;
	}
	cutilSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(rigid_SPH_massD), &rigid_SPH_mass, sizeof(rigid_SPH_mass)));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(startRigidMarkersD), &startRigidMarkers, sizeof(startRigidMarkers)); //can be defined outside of the kernel, and only once
	hipMemcpyToSymbolAsync(HIP_SYMBOL(numRigid_SphMarkersD), &numRigid_SphMarkers, sizeof(numRigid_SphMarkers)); //can be defined outside of the kernel, and only once

	printf("a7 yoho\n");

		//******************************************************************************
	thrust::device_vector<real3> rigidSPH_MeshPos_LRF_D(numRigid_SphMarkers);
	uint nBlocks_numRigid_SphMarkers;
	uint nThreads_SphMarkers;
	computeGridSize(numRigid_SphMarkers, 256, nBlocks_numRigid_SphMarkers, nThreads_SphMarkers);
	printf("before first kernel\n");
	Populate_RigidSPH_MeshPos_LRF_kernel<<<nBlocks_numRigid_SphMarkers, nThreads_SphMarkers>>>(R3CAST(rigidSPH_MeshPos_LRF_D), R3CAST(posRadD), I1CAST(rigidIdentifierD), R3CAST(posRigidD), startRigidMarkers, numRigid_SphMarkers);
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: CalcTorqueShare");	printf("after first kernel\n");

	//******************************************************************************
	//******************** flex body some initialization

	int numFlBcRigid = 2 + numRigidBodies;
	int numFlexBodies = ANCF_Beam_Length.size();
//	int totalNumberOfFlexNodes = ANCF_ReferenceArrayNodesOnBeams[ANCF_ReferenceArrayNodesOnBeams.size() - 1].y;

	int startFlexMarkers = (referenceArray[numFlBcRigid-1]).y;
	int numFlex_SphMarkers = referenceArray[numFlBcRigid + numFlexBodies - 1].y - startFlexMarkers;
	hipMemcpyToSymbolAsync(HIP_SYMBOL(startFlexMarkersD), &startFlexMarkers, sizeof(startFlexMarkers)); //can be defined outside of the kernel, and only once
	hipMemcpyToSymbolAsync(HIP_SYMBOL(numFlex_SphMarkersD), &numFlex_SphMarkers, sizeof(numFlex_SphMarkers)); //can be defined outside of the kernel, and only once

		//******************************************************************************
	thrust::device_vector<real3> flexSPH_MeshPos_LRF_D(numFlex_SphMarkers);
	thrust::device_vector<real3> flexSPH_MeshSlope_Initial_D(numFlex_SphMarkers);
	uint nBlocks_numFlex_SphMarkers;
	uint nThreads_SphMarkers;
	computeGridSize(numFlex_SphMarkers, 256, nBlocks_numFlex_SphMarkers, nThreads_SphMarkers);
	printf("before first kernel\n");

	Populate_FlexSPH_MeshPos_LRF_kernel<<<nBlocks_numFlex_SphMarkers, nThreads_SphMarkers>>>(R3CAST(flexSPH_MeshPos_LRF_D), R3CAST(posRadD), I1CAST(flexIdentifierD), R1CAST(parametricDist), R1CAST(ANCF_Beam_Length),
			I1CAST(ANCF_NumNodes_Per_Beam), R3CAST(ANCF_Nodes), R3CAST(ANCF_Slopes));
	hipDeviceSynchronize();
		CUT_CHECK_ERROR("Kernel execution failed: Populate_FlexSPH_MeshPos_LRF_kernel");	printf("after first kernel\n");

	Populate_FlexSPH_MeshSlope_LRF_kernel<<<nBlocks_numFlex_SphMarkers, nThreads_SphMarkers>>>(R3CAST(flexSPH_MeshSlope_Initial_D), I1CAST(flexIdentifierD), R1CAST(parametricDist), R1CAST(ANCF_Beam_Length),
				I1CAST(ANCF_NumNodes_Per_Beam), R3CAST(ANCF_Nodes), R3CAST(ANCF_Slopes));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: Populate_FlexSPH_MeshSlope_LRF_kernel");	printf("after first kernel\n");

	//******************************************************************************
	thrust::device_vector<real4> qD1 = mQuatRot;
	thrust::device_vector<real3> AD1(numRigidBodies);
	thrust::device_vector<real3> AD2(numRigidBodies);
	thrust::device_vector<real3> AD3(numRigidBodies);
	uint nBlock_UpdateRigid;
	uint nThreads_rigidParticles;
	computeGridSize(numRigidBodies, 128, nBlock_UpdateRigid, nThreads_rigidParticles);
	RotationMatirixFromQuaternion_kernel<<<nBlock_UpdateRigid, nThreads_rigidParticles>>>(R3CAST(AD1), R3CAST(AD2), R3CAST(AD3), R4CAST(qD1));
	hipDeviceSynchronize();
	CUT_CHECK_ERROR("Kernel execution failed: UpdateRotation");

	//int i =  rigidIdentifierD[429];
	//printf("rigid body coord %d %f %f\n", i, posRigidH[i].x, posRigidH[i].z);
	//printf("length %f\n", length(R2(posRigidH[i].x - .003474, posRigidH[i].z - .000673)));

	//****************************** bin size adjustement and contact detection stuff *****************************
	//real_ mBinSize0 = (mNSpheres == 0) ? mBinSize0 : 2 * HSML;
	//real3 cMinOffsetCollisionPurpose = cMin - 3 * R3(0, mBinSize0, mBinSize0);		//periodic bc in x direction
	//real3 cMaxOffsetCollisionPurpose = cMax + 3 * R3(0, mBinSize0, mBinSize0);
	////real3 cMinOffsetCollisionPurpose = cMin - 3 * R3(mBinSize0, mBinSize0, mBinSize0);		//periodic bc in x direction
	////real3 cMaxOffsetCollisionPurpose = cMax + 3 * R3(mBinSize0, mBinSize0, mBinSize0);

	/////printf("side.x %f\n", abs(cMaxOffsetCollisionPurpose.x - cMinOffsetCollisionPurpose.x) / mBinSize);
	//int3 SIDE = I3(  floor( (cMaxOffsetCollisionPurpose.x - cMinOffsetCollisionPurpose.x) / mBinSize0 ), floor( (cMaxOffsetCollisionPurpose.y - cMinOffsetCollisionPurpose.y) / mBinSize0 ), floor( (cMaxOffsetCollisionPurpose.z - cMinOffsetCollisionPurpose.z) / mBinSize0)  );
	//real_ mBinSize = (cMaxOffsetCollisionPurpose.x - cMinOffsetCollisionPurpose.x) / SIDE.x;  //this one works when periodic BC is only on x. if it was on y as well (or on z), you would have problem.
	real3 cMinOffsetCollisionPurpose = cMin;// - 3 * R3(0, 0, binSize0); //periodic bc in x direction
	real3 cMaxOffsetCollisionPurpose = cMax;// + 3 * R3(0, 0, binSize0);
	int3 SIDE = I3(int((cMaxOffsetCollisionPurpose.x - cMinOffsetCollisionPurpose.x) / binSize0 + .1), int((cMaxOffsetCollisionPurpose.y - cMinOffsetCollisionPurpose.y) / binSize0 + .1),
			floor((cMaxOffsetCollisionPurpose.z - cMinOffsetCollisionPurpose.z) / binSize0 + .1));
	real_ mBinSize = binSize0; //Best solution in that case may be to change cMax or cMin such that periodic sides be a multiple of binSize

	printf("SIDE: %d, %d, %d\n", SIDE.x, SIDE.y, SIDE.z);
	//*******************
	SimParams paramsH;
	paramsH.gravity = Gravity; //Gravity * sizeScale;;// R3(0, -9.8, 0) * sizeScale; //R3(0, -9800, 0) * sizeScale;
	paramsH.particleRadius = HSML;
	paramsH.gridSize = SIDE;
	//paramsH.numCells = SIDE.x * SIDE.y * SIDE.z;
	paramsH.worldOrigin = cMinOffsetCollisionPurpose;
	paramsH.cellSize = R3(mBinSize, mBinSize, mBinSize);
	paramsH.boxDims = cMaxOffsetCollisionPurpose - cMinOffsetCollisionPurpose;
	printf("boxDims: %f, %f, %f\n", paramsH.boxDims.x, paramsH.boxDims.y, paramsH.boxDims.z);

	setParameters(&paramsH);
	cutilSafeCall( hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), &paramsH, sizeof(SimParams)));

	//********************************************************************************
	int numberOfSections = 20; //number of sections for measuring the distribution
	thrust::device_vector<int>  distributionD(numberOfSections);

	FILE *outFileMultipleZones;

	int povRayCounter = 0;
	int stepEnd = 1.0e6;//2.4e6;//600000;//2.4e6 * (.02 * sizeScale) / delT ; //1.4e6 * (.02 * sizeScale) / delT ;//0.7e6 * (.02 * sizeScale) / delT ;//0.7e6;//2.5e6; //200000;//10000;//50000;//100000;
	printf("stepEnd %d\n", stepEnd);

	real_ delTOrig = delT;
	//for (int tStep = 0; tStep < 0; tStep ++) {
	for (int tStep = 0; tStep < stepEnd + 1; tStep++) {
		//edit  since yu deleted cyliderRotOmegaJD

//		if (tStep > 10000) delT = .2;
		GpuTimer myGpuTimer;
		myGpuTimer.Start();

		if (tStep < 1000) delT = 0.25 * delTOrig; else delT = delTOrig;
		//computations
		thrust::device_vector<real3> posRadD2 = posRadD;
		thrust::device_vector<real4> velMasD2 = velMasD;
		thrust::device_vector<real4> rhoPresMuD2 = rhoPresMuD;
		thrust::device_vector<real3> posRigidD2 = posRigidD;
		thrust::device_vector<real3> posRadRigidCumulativeD2 = posRigidCumulativeD;
		thrust::device_vector<real4> velMassRigidD2 = velMassRigidD;
		thrust::device_vector<real3> omegaLRF_D2 = omegaLRF_D;
		thrust::device_vector<real3> vel_XSPH_D(mNSpheres);
		thrust::device_vector<real3> AD1_2 = AD1;
		thrust::device_vector<real3> AD2_2 = AD2;
		thrust::device_vector<real3> AD3_2 = AD3;
		thrust::device_vector<real4> qD2 = qD1;

		ForceSPH(posRadD, velMasD, vel_XSPH_D, rhoPresMuD, bodyIndexD, derivVelRhoD, referenceArray, mNSpheres, SIDE, 0.5 * delT); //?$ right now, it does not consider gravity or other stuff on rigid bodies. they should be applied at rigid body solver
		UpdateFluid(posRadD2, velMasD2, vel_XSPH_D, rhoPresMuD2, derivVelRhoD, referenceArray, 0.5 * delT); //assumes ...D2 is a copy of ...D
		//UpdateBoundary(posRadD2, velMasD2, rhoPresMuD2, derivVelRhoD, referenceArray, 0.5 * delT);		//assumes ...D2 is a copy of ...D
		UpdateRigidBody(posRadD2, velMasD2, posRigidD2, posRadRigidCumulativeD2, velMassRigidD2, qD2, AD1_2, AD2_2, AD3_2, omegaLRF_D2, derivVelRhoD, rigidIdentifierD,
				rigidSPH_MeshPos_LRF_D, referenceArray, jD1, jD2, jInvD1, jInvD2, paramsH, numRigidBodies, startRigidMarkers, numRigid_SphMarkers, float(tStep)/stepEnd, 0.5 * delT);
		ApplyBoundary(posRadD2, rhoPresMuD2, mNSpheres, posRigidD2, velMassRigidD2, numRigidBodies);

		ForceSPH(posRadD2, velMasD2, vel_XSPH_D, rhoPresMuD2, bodyIndexD, derivVelRhoD, referenceArray, mNSpheres, SIDE, delT);
		UpdateFluid(posRadD, velMasD, vel_XSPH_D, rhoPresMuD, derivVelRhoD, referenceArray, delT);
		//UpdateBoundary(posRadD, velMasD, rhoPresMuD, derivVelRhoD, referenceArray, delT);
		UpdateRigidBody(posRadD, velMasD, posRigidD, posRigidCumulativeD, velMassRigidD, qD1, AD1, AD2, AD3, omegaLRF_D, derivVelRhoD, rigidIdentifierD,
				rigidSPH_MeshPos_LRF_D, referenceArray, jD1, jD2, jInvD1, jInvD2, paramsH, numRigidBodies, startRigidMarkers, numRigid_SphMarkers, float(tStep)/stepEnd, delT);
//			/* post_process for Segre-Silberberg */
//			if(tStep >= 0) {
//				real2 channelCenter = .5 * R2(cMax.y + cMin.y, cMax.z + cMin.z);
//				FindPassesFromTheEnd(posRigidD, distributionD, numRigidBodies, channelCenter, channelRadius, numberOfSections);
//			}
		ApplyBoundary(posRadD, rhoPresMuD, mNSpheres, posRigidD, velMassRigidD, numRigidBodies);

		posRadD2.clear();
		velMasD2.clear();
		rhoPresMuD2.clear();
		posRigidD2.clear();
		posRadRigidCumulativeD2.clear();
		velMassRigidD2.clear();
		vel_XSPH_D.clear();
		qD2.clear();
		AD1_2.clear();
		AD2_2.clear();
		AD3_2.clear();
		omegaLRF_D2.clear();

		//density re-initialization
		if (tStep % 10 == 0) {
			DensityReinitialization(posRadD, velMasD, rhoPresMuD, mNSpheres, SIDE); //does not work for analytical boundaries (non-meshed) and free surfaces
		}

		//************************************************
		//edit  since yu deleted cyliderRotOmegaJD
		PrintToFile(posRadD, velMasD, rhoPresMuD, referenceArray, rigidIdentifierD, posRigidD, posRigidCumulativeD, velMassRigidD, qD1, AD1, AD2, AD3, omegaLRF_D, cMax, cMin, paramsH,
				delT, tStep, channelRadius, channelCenterYZ);

//		PrintToFileDistribution(distributionD, channelRadius, numberOfSections, tStep);
		//************
		myGpuTimer.Stop();
		real_ time2 = (real_)myGpuTimer.Elapsed();
		if (tStep % 50 == 0) {
			printf("step: %d, step Time: %f\n ", tStep, time2);
			//printf("a \n");
		}
		fflush(stdout);

		//_CrtDumpMemoryLeaks(); //for memory leak detection (msdn suggestion for VS) apparently does not work in conjunction with cuda

	}

	//you may copy back to host
	posRadD.clear();
	velMasD.clear();
	rhoPresMuD.clear();
	posRigidD.clear();
	posRigidCumulativeD.clear();
	velMassRigidD.clear();
	omegaLRF_D.clear();
	bodyIndexD.clear();
	derivVelRhoD.clear();
	rigidIdentifierD.clear();
	rigidSPH_MeshPos_LRF_D.clear();
	flexSPH_MeshPos_LRF_D.clear();
	flexSPH_MeshSlope_Initial_D.clear();
	qD1.clear();
	AD1.clear();
	AD2.clear();
	AD3.clear();
	distributionD.clear();

	jD1.clear();
	jD2.clear();
	jInvD1.clear();
	jInvD2.clear();

	myTotalTime.Stop();
	real_ time = (real_)myTotalTime.Elapsed();
	printf("total Time: %f\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n ", time);
}
