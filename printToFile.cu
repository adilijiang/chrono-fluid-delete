#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <sstream>
#include <fstream>
#include <thrust/reduce.h>
#include "printToFile.cuh"
#include "custom_cutil_math.h"
#include "SPHCudaUtils.h"
using namespace std;


real_ AngleF3F3(real3 a, real3 b) {
	return acos(dot(a, b));
}
//*******************************************************************************************************************************
//builds the neighbors' list of each particle and finds the force on each particle
//calculates the interaction force between 1- fluid-fluid, 2- fluid-solid, 3- solid-fluid particles
//calculates forces from other SPH or solid particles, as wall as boundaries
void PrintToFile(
		thrust::device_vector<real3> & posRadD,
		thrust::device_vector<real4> & velMasD,
		thrust::device_vector<real4> & rhoPresMuD,
		const thrust::host_vector<int3> & referenceArray,
		const thrust::device_vector<int> & rigidIdentifierD,
		thrust::device_vector<real3> & posRigidD,
		thrust::device_vector<real3> & posRigidCumulativeD,
		thrust::device_vector<real4> & velMassRigidD,
		thrust::device_vector<real4> & qD1,
		thrust::device_vector<real3> & AD1,
		thrust::device_vector<real3> & AD2,
		thrust::device_vector<real3> & AD3,
		thrust::device_vector<real3> & omegaLRF_D,
		real3 cMax,
		real3 cMin,
		SimParams paramsH,
		real_ delT,
		int tStep,
		real_ channelRadius,
		real2 channelCenterYZ) {
	thrust::host_vector<real3> posRadH = posRadD;
	thrust::host_vector<real4> velMasH = velMasD;
	thrust::host_vector<real4> rhoPresMuH = rhoPresMuD;
	thrust::host_vector<int> rigidIdentifierH = rigidIdentifierD;
	thrust::host_vector<real3> posRigidH = posRigidD;
	thrust::host_vector<real3> posRigidCumulativeH = posRigidCumulativeD;
	thrust::host_vector<real4> velMassRigidH = velMassRigidD;
	thrust::host_vector<real4> qH1 = qD1;
	thrust::host_vector<real3> omegaLRF_H = omegaLRF_D;
//////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++com
	ofstream fileNameFluid;
	int stepSaveFluid = 200000;
	if (tStep % stepSaveFluid == 0) {
		if (tStep / stepSaveFluid == 0) {
			fileNameFluid.open("dataFluid.txt");
			fileNameFluid<<"variables = \"x\", \"y\", \"z\", \"Vx\", \"Vy\", \"Vz\", \"Velocity Magnitude\", \"Rho\", \"Pressure\", \"type\"\n";
		} else {
			fileNameFluid.open("dataFluid.txt", ios::app);
		}

		fileNameFluid<<"zone\n";
		stringstream ssFluid;
		for (int i = referenceArray[0].x; i < referenceArray[1].y; i++) {
			real3 pos = posRadH[i];
			real3 vel = R3(velMasH[i]);
			real4 rP = rhoPresMuH[i];
			real_ velMag = length(vel);
			ssFluid<< pos.x<<", "<< pos.y<<", "<< pos.z<<", "<< vel.x<<", "<< vel.y<<", "<< vel.z<<", "<< velMag<<", "<< rP.x<<", "<< rP.y<<", "<< rP.w << endl;
		}
		fileNameFluid<<ssFluid.str();
		fileNameFluid.close();
	}
////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	ofstream fileNameRigidsSPH;
	int stepSaveRigid = 10000;
	///if (tStep % 20 == 0 && tStep > 56000) {
	//if (tStep > 12506) {
	if (tStep % stepSaveRigid == 0) {
		if (tStep / stepSaveRigid == 0) {
			fileNameRigidsSPH.open("dataRigidParticle.txt");
			fileNameRigidsSPH<<"variables = \"x\", \"y\", \"z\", \"Vx\", \"Vy\", \"Vz\", \"Velocity Magnitude\", \"OmegaX\", \"OmegaY\", \"OmegaZ\", \"Rho\", \"Pressure\", \"bodySize\", \"type\"\n";

//			fprintf(
//					fileNameRigidsSPH,
//					);
		} else {
			fileNameRigidsSPH.open("dataRigidParticle.txt", ios::app);
		}
		fileNameRigidsSPH<<"zone\n";
		//fprintf(fileNameRigidsSPH, "zone\n");
		stringstream ssRigidsSPH;
		if (referenceArray.size() > 2) {
			const int numRigidBodies = posRigidH.size();
			int startRigidParticle = (referenceArray[2]).x;

			for (int i = startRigidParticle; i < referenceArray[2 + numRigidBodies - 1].y; i++) {
				real3 pos = posRadH[i];
				real3 vel = R3(velMasH[i]);
				//printf("velocccc %f %f %f\n", vel.x, vel.y, vel.z);
				real4 rP = rhoPresMuH[i];
				real_ velMag = length(vel);
				int rigidID = rigidIdentifierH[i - startRigidParticle];
				real3 posRigid = posRigidH[rigidID];
				real3 omega = omegaLRF_H[rigidID];
				real_ fakeRad = 9;
				ssRigidsSPH<<pos.x<<", "<< pos.y<<", "<< pos.z<<", "<<vel.x<<", "<<vel.y<<", "<< vel.z<<", "<< velMag<<", "<<omega.x<<", "<< omega.y<<", "<< omega.z<<", "<< rP.x<<", "<< rP.y<<", "<<fakeRad<<", "<< rP.w<<endl;

//				fprintf(fileNameRigidsSPH, "%f, %f, %f, %f, %f, %f, %f, %f, %f, %f, %f, %f, %f, %f\n", pos.x, pos.y, pos.z, vel.x, vel.y, vel.z, velMag,
//						omega.x, omega.y, omega.z, rP.x, rP.y, fakeRad, rP.w);
			}
		}
		fileNameRigidsSPH<<ssRigidsSPH.str();
		fileNameRigidsSPH.close();
	//	fflush(fileNameRigidsSPH);
		//fclose(fileNameRigidsSPH);
	}
//////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	ofstream fileNameSlice;
	int stepSaveFluidSlice = 50000; //1;//20000;
	//if (tStep%100 == 0 &&  tStep > 20400) {
	//if (tStep > 49100) {
	if (tStep % stepSaveFluidSlice == 0) {
		//if (tStep / stepSaveFluidSlice == 49101) {
		if (tStep / stepSaveFluidSlice == 0) {
			fileNameSlice.open("dataTotalSlice.txt");
			fileNameSlice<<"variables = \"x\", \"y\", \"z\", \"Vx\", \"Vy\", \"Vz\", \"Velocity Magnitude\", \"Rho\", \"Pressure\", \"type\"\n";
		} else {
			fileNameSlice.open("dataTotalSlice.txt", ios::app);
		}
		fileNameSlice<<"zone\n";
		stringstream ssSlice;
		for (int i = referenceArray[0].x; i < referenceArray[referenceArray.size() - 1].y; i++) {
			real3 posRad = posRadH[i];
			real3 pos = posRad;
			real_ rad = HSML;
			real3 vel = R3(velMasH[i]);
			real4 rP = rhoPresMuH[i];
			real_ velMag = length(vel);
			if ((pos.y < cMin.y + 0.5 * (cMax.y - cMin.y) + 3 * rad) && (pos.y > cMin.y + 0.5 * (cMax.y - cMin.y) - 3 * rad)) {
				ssSlice<< pos.x<<", "<< pos.y<<", "<< pos.z<<", "<< vel.x<<", "<< vel.y<<", "<< vel.z<<", "<< velMag<<", "<< rP.x<<", "<< rP.y<<", "<< rP.w<<endl;
			}
		}
		fileNameSlice<<ssSlice.str();
		fileNameSlice.close();
	}
////////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++//comcom
//	ofstream fileNameCartesianTotal;
//	thrust::host_vector<real4> rho_Pres_CartH(1);
//	thrust::host_vector<real4> vel_VelMag_CartH(1);
//	real_ resolution = 2 * HSML;
//	int3 cartesianGridDims;
//	int tStepCartesianTotal = 1000000;
//	int tStepCartesianSlice = 100000;
//	int tStepPoiseuilleProf = 1000; //tStepCartesianSlice;
//
//	int stepCalcCartesian = min(tStepCartesianTotal, tStepCartesianSlice);
//	stepCalcCartesian = min(stepCalcCartesian, tStepPoiseuilleProf);
//
//	if (tStep % stepCalcCartesian == 0) {
//		MapSPH_ToGrid(resolution, cartesianGridDims, rho_Pres_CartH, vel_VelMag_CartH, posRadD, velMasD, rhoPresMuD,
//				referenceArray[referenceArray.size() - 1].y, paramsH);
//	}
//	if (tStep % tStepCartesianTotal == 0) {
//		if (tStep / tStepCartesianTotal == 0) {
//			fileNameCartesianTotal.open("dataCartesianTotal.txt");
//			fileNameCartesianTotal<<"variables = \"x\", \"y\", \"z\", \"Vx\", \"Vy\", \"Vz\", \"Velocity Magnitude\", \"Rho\", \"Pressure\"\n";
//		} else {
//			fileNameCartesianTotal .open("dataCartesianTotal.txt", ios::app);
//		}
//		fileNameCartesianTotal<<"zone I = "<<cartesianGridDims.x<<", J = "<<cartesianGridDims.y<<", K = "<<cartesianGridDims.z<<endl;
//		stringstream ssCartesianTotal;
//		for (int k = 0; k < cartesianGridDims.z; k++) {
//			for (int j = 0; j < cartesianGridDims.y; j++) {
//				for (int i = 0; i < cartesianGridDims.x; i++) {
//					int index = i + j * cartesianGridDims.x + k * cartesianGridDims.x * cartesianGridDims.y;
//					real3 gridNodeLoc = resolution * R3(i, j, k) + paramsH.worldOrigin;
//					ssCartesianTotal<<gridNodeLoc.x<<", "<< gridNodeLoc.y<<", "<< gridNodeLoc.z<<", "<<
//							vel_VelMag_CartH[index].x<<", "<< vel_VelMag_CartH[index].y<<", "<< vel_VelMag_CartH[index].z<<", "<< vel_VelMag_CartH[index].w<<", "<<
//							rho_Pres_CartH[index].x<<", "<< rho_Pres_CartH[index].y<<endl;
//				}
//			}
//		}
//		fileNameCartesianTotal<<ssCartesianTotal.str();
//		fileNameCartesianTotal.close();
//	}
//////////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ //comcom
//	ofstream fileNameCartesianMidplane;
//	if (tStep % tStepCartesianSlice == 0) {
//		if (tStep / tStepCartesianSlice == 0) {
//			fileNameCartesianMidplane.open("dataCartesianMidplane.txt");
//			fileNameCartesianMidplane<<"variables = \"x\", \"z\", \"Vx\", \"Vy\", \"Vz\", \"Velocity Magnitude\", \"Rho\", \"Pressure\"\n";
//		} else {
//			fileNameCartesianMidplane .open("dataCartesianMidplane.txt", ios::app);
//		}
//		fileNameCartesianMidplane<< "zone I = "<<cartesianGridDims.x<<", J = "<<cartesianGridDims.z<<"\n";
//		int j = cartesianGridDims.y / 2;
//		stringstream ssCartesianMidplane;
//		for (int k = 0; k < cartesianGridDims.z; k++) {
//			for (int i = 0; i < cartesianGridDims.x; i++) {
//				int index = i + j * cartesianGridDims.x + k * cartesianGridDims.x * cartesianGridDims.y;
//				real3 gridNodeLoc = resolution * R3(i, j, k) + paramsH.worldOrigin;
//				ssCartesianMidplane<<gridNodeLoc.x<<", "<< gridNodeLoc.z<<", "<< vel_VelMag_CartH[index].x<<", "<<
//						vel_VelMag_CartH[index].y<<", "<< vel_VelMag_CartH[index].z<<", "<< vel_VelMag_CartH[index].w<<", "<< rho_Pres_CartH[index].x<<", "<<
//						rho_Pres_CartH[index].y<<endl;
//			}
//		}
//		fileNameCartesianMidplane<<ssCartesianMidplane.str();
//		fileNameCartesianMidplane.close();
//	}
//	rho_Pres_CartH.clear();
//////////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++comcom
//	ofstream fileVelocityProfPoiseuille;
//	if (tStep % tStepPoiseuilleProf == 0) {
//		if (tStep / tStepPoiseuilleProf == 0) {
//			fileVelocityProfPoiseuille.open("dataVelProfile.txt");
//			fileVelocityProfPoiseuille<< "variables = \"Z(m)\", \"Vx(m/s)\"\n";
//
//		} else {
//			fileVelocityProfPoiseuille.open("dataVelProfile.txt", ios::app);
//		}
//		fileVelocityProfPoiseuille<<"zone T=\"t = "<<delT * tStep<<"\""endl;
//		stringstream ssVelocityProfPoiseuille;
//		int j = cartesianGridDims.y / 2;
//		int i = cartesianGridDims.x / 2;
//		for (int k = 0; k < cartesianGridDims.z; k++) {
//			int index = i + j * cartesianGridDims.x + k * cartesianGridDims.x * cartesianGridDims.y;
//			real3 gridNodeLoc = resolution * R3(i, j, k) + paramsH.worldOrigin;
//			if (gridNodeLoc.z > 1 * sizeScale && gridNodeLoc.z < 2 * sizeScale) {
//				ssVelocityProfPoiseuille<<gridNodeLoc.z<<", "<< vel_VelMag_CartH[index].x<<endl;
//			}
//		}
//		fileVelocityProfPoiseuille<<ssVelocityProfPoiseuille.str();
//		fileVelocityProfPoiseuille.close();
//	}
//	vel_VelMag_CartH.clear();
//////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
	ofstream fileRigidParticleCenterVsTimeAndDistance;
	int numRigidBodiesInOnePeriod = int(posRigidH.size() / real_(nPeriod) + .5);
	int tStepRigidCenterPos = 1000;
	if (tStep % tStepRigidCenterPos == 0) {
		if (tStep / tStepRigidCenterPos == 0) {
			fileRigidParticleCenterVsTimeAndDistance.open("dataRigidCenterVsTimeAndDistance.txt");
			fileRigidParticleCenterVsTimeAndDistance<<"(t, x, dist[or y], dum[or z],   x_cumul, y, z,   vx, vy, vz,   omega_x, omega_y, omega_z) (sequentially for all particles), average flow velocity (x, y, z, magnitude), channel radius\n" << endl;
		} else {
			fileRigidParticleCenterVsTimeAndDistance.open("dataRigidCenterVsTimeAndDistance.txt", ios::app);
		}
//		(void) thrust::reduce_by_key(rigidIdentifierD.begin(), rigidIdentifierD.end(), torqueParticlesD.begin(), dummyIdentify.begin(),
//					totalTorque3.begin(), binary_pred, thrust::plus<real3>());

		printf("channel radius %f\n", channelRadius);
		real4 sumVelocity = R4(0);
		real4 initSumR4 = R4(0);
		sumVelocity = thrust::reduce(velMasD.begin() + referenceArray[0].x, velMasD.begin() + referenceArray[0].y, initSumR4, thrust::plus<real4>());
		real3 aveVel = R3(sumVelocity / (referenceArray[0].y - referenceArray[0].x));

		stringstream ssParticleCenterVsTime;
		if (referenceArray.size() > 2) {
			for (int j = 0; j < numRigidBodiesInOnePeriod; j++) {
				real3 p_rigid = posRigidH[j];
				real3 v_rigid = R3(velMassRigidH[j]);
				real3 omega_rigid = omegaLRF_H[j];
				//printf("position %f %f %f %f\n", p_rigid.x, p_rigid.y, p_rigid.z,0);
				real3 p_rigidCumul = posRigidCumulativeH[j];
//				//***cartesian distance (channel, duct)
//				ssParticleCenterVsTime << tStep * delT << ", " << p_rigid.y << ", " << p_rigid.z;
//				ssParticleCenterVsDistance << p_rigidCumul.x << ", " << p_rigid.y << ", " << p_rigid.z;

//				//***radial distance (tube)
				real2 dist2 = R2(channelCenterYZ.x - p_rigid.y, channelCenterYZ.y - p_rigid.z);
				printf("center %f %f and radius %f and py and pz %f %f\n", channelCenterYZ.x, channelCenterYZ.y, channelRadius, p_rigid.y, p_rigid.z);
				ssParticleCenterVsTime << tStep * delT << ", " <<  p_rigidCumul.x << ", " <<
						length(dist2) / channelRadius << ", " << length(dist2) / channelRadius << ", " <<
						p_rigidCumul.x << ", " << p_rigid.y << ", " << p_rigid.z << ", " <<
						v_rigid.x << ", " << v_rigid.y << ", " << v_rigid.z << ", " <<
						omega_rigid.x << ", " << omega_rigid.y << ", " << omega_rigid.z << ", " << aveVel.x << ", " << aveVel.y << ", " << aveVel.z << ", " << length(aveVel) << ", " << channelRadius << ", ";
			}
			ssParticleCenterVsTime<<endl;
			//fprintf(fileRigidParticleCenterVsTimeAndDistance, "\n");
			//fprintf(fileRigidParticleCenterVsDistance, "\n");
		}
		fileRigidParticleCenterVsTimeAndDistance << ssParticleCenterVsTime.str();
		fileRigidParticleCenterVsTimeAndDistance.close();
	}
//////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
		ofstream fileRigidParticlesDataForTecplot;
		int tStepRigidParticlesDataForTecplot = 1000;
		if (tStep % tStepRigidParticlesDataForTecplot == 0) {
			if (tStep / tStepRigidParticlesDataForTecplot == 0) {
				fileRigidParticlesDataForTecplot.open("dataRigidParticlesDataForTecplot.txt");
				fileRigidParticlesDataForTecplot<<"PipeRadius, PipeLength\n";
				fileRigidParticlesDataForTecplot<<	channelRadius <<", "<< cMax.x - cMin.x<<", "<< posRigidH.size()<<endl;
				fileRigidParticlesDataForTecplot<<"variables = \"t(s)\", \"x\", \"y\", \"z\", \"r\", \"CumulX\", \"vX\", \"vY\", \"vZ\", \"velMagnitude\", \"angleAxisXWithPipeAxis\", \"angleAxisYWithPipeAxis\", \"angleAxisZWithPipeAxis\"\n";
			} else {
				fileRigidParticlesDataForTecplot.open("dataRigidParticlesDataForTecplot.txt", ios::app);
			}
			fileRigidParticlesDataForTecplot<<"zone\n";
			stringstream ssRigidParticlesDataForTecplot;
			for (int j = 0; j < posRigidH.size(); j++) {
				real3 p_rigid = posRigidH[j];

				//rotate the principal axis
				real3 aD1 = AD1[j];
				real3 aD2 = AD2[j];
				real3 aD3 = AD3[j];
				real3 axisX = R3(aD1.x, aD2.x, aD3.x);
				real3 axisY = R3(aD1.y, aD2.y, aD3.y);
				real3 axisZ = R3(aD1.z, aD2.z, aD3.z);

				real4 q_rigid = qH1[j];
				real3 p_rigidCumul = posRigidCumulativeH[j];
				real3 v_rigid = R3(velMassRigidH[j]);
				real2 dist2 = R2(channelCenterYZ.x - p_rigid.y, channelCenterYZ.y - p_rigid.z);
				real_ angleAxisXWithPipeAxis = AngleF3F3(axisX, R3(1, 0, 0));
				real_ angleAxisYWithPipeAxis = AngleF3F3(axisY, R3(1, 0, 0));
				real_ angleAxisZWithPipeAxis = AngleF3F3(axisZ, R3(1, 0, 0));

				ssRigidParticlesDataForTecplot<<tStep * delT<<", "<<p_rigid.x<<", "<<p_rigid.y<<", "<<p_rigid.z<<", "<<length(dist2) / channelRadius<<", "<<p_rigidCumul.x<<", "<<v_rigid.x<<", "<<v_rigid.y<<", "<<v_rigid.z<<", "<<length(v_rigid)<<", "<<angleAxisXWithPipeAxis<<", "<<angleAxisYWithPipeAxis<<", "<<angleAxisZWithPipeAxis<<endl;
			}
			fileRigidParticlesDataForTecplot << ssRigidParticlesDataForTecplot.str();
			fileRigidParticlesDataForTecplot.close();
		}
//////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++com
		ofstream fileNameRigidBodies;
		ofstream fileNameFluidParticles;
		ofstream fileNameBoundaries;
		ofstream fileNameFluidBoundaries;

		system("mkdir -p povFiles");
		int tStepsPovFiles = 2000;
		if (tStep % tStepsPovFiles == 0) {
			if (tStep / tStepsPovFiles == 0) {
						//linux. In windows, it is System instead of system (to invoke a command in the command line)
				system("rm povFiles/*.csv");
			}
			char fileCounter[5];
			int dumNumChar = sprintf(fileCounter, "%d", int(tStep / tStepsPovFiles) );

			char nameRigid[255];
			sprintf(nameRigid, "povFiles/rigid");
			strcat(nameRigid, fileCounter);
			strcat(nameRigid, ".csv");
			char nameFluid[255];
			sprintf(nameFluid, "povFiles/fluid");
			strcat(nameFluid, fileCounter);
			strcat(nameFluid, ".csv");
			char nameBoundary[255];
			sprintf(nameBoundary, "povFiles/boundary");
			strcat(nameBoundary, fileCounter);
			strcat(nameBoundary, ".csv");
			char nameFluidBoundaries[255];
			sprintf(nameFluidBoundaries, "povFiles/fluid_boundary");
			strcat(nameFluidBoundaries, fileCounter);
			strcat(nameFluidBoundaries, ".csv");

			fileNameRigidBodies.open(nameRigid);
			stringstream ssRigidBodies;
			if (referenceArray.size() > 2) {
				const int numRigidBodies = posRigidH.size();
				for (int j = 0; j < numRigidBodies; j++) {
					real3 p_rigid = posRigidH[j];
					real4 q_rigid = qH1[j];
					real4 velMassRigid = velMassRigidH[j];
					ssRigidBodies<<tStep * delT<<", "<< p_rigid.x<<", "<< p_rigid.y<<", "<< p_rigid.z<<", "<< velMassRigid.x << ", " <<  velMassRigid.y << ", " <<  velMassRigid.z <<", "<< length(R3(velMassRigid)) <<  ", "<< length(R3(velMassRigid)) << ", "<< q_rigid.x<<", "<< q_rigid.y<<", "<< q_rigid.z<<", "<< q_rigid.w<<", "<<endl;
				}
			}
			fileNameRigidBodies << ssRigidBodies.str();
			fileNameRigidBodies.close();

			fileNameFluidParticles.open(nameFluid);
			stringstream ssFluidParticles;
			for (int i = referenceArray[0].x; i < referenceArray[0].y; i++) {
				real3 pos = posRadH[i];
				real3 vel = R3(velMasH[i]);
				real4 rP = rhoPresMuH[i];
				real_ velMag = length(vel);
				ssFluidParticles<< pos.x<<", "<< pos.y<<", "<< pos.z<<", "<< vel.x<<", "<< vel.y<<", "<< vel.z<<", "<< velMag<<", "<< rP.x<<", "<< rP.y<<", "<< rP.w<<", "<<endl;
			}
			fileNameFluidParticles<<ssFluidParticles.str();
			fileNameFluidParticles.close();

			fileNameBoundaries.open(nameBoundary);
			stringstream ssBoundary;
			for (int i = referenceArray[1].x; i < referenceArray[1].y; i++) {
				real3 pos = posRadH[i];
				real3 vel = R3(velMasH[i]);
				real4 rP = rhoPresMuH[i];
				real_ velMag = length(vel);
				ssBoundary<<pos.x<<", "<< pos.y<<", "<< pos.z<<", "<< vel.x<<", "<< vel.y<<", "<< vel.z<<", "<< velMag<<", "<< rP.x<<", "<< rP.y<<", "<< rP.w<<", "<<endl;
			}
			fileNameBoundaries << ssBoundary.str();
			fileNameBoundaries.close();

			fileNameFluidBoundaries.open(nameFluidBoundaries);
			stringstream ssFluidBoundaryParticles;
	//		ssFluidBoundaryParticles.precision(20);
			for (int i = referenceArray[0].x; i < referenceArray[1].y; i++) {
				real3 pos = posRadH[i];
				real3 vel = R3(velMasH[i]);
				real4 rP = rhoPresMuH[i];
				real_ velMag = length(vel);
				ssFluidBoundaryParticles<< pos.x<<", "<< pos.y<<", "<< pos.z<<", "<< vel.x<<", "<< vel.y<<", "<< vel.z<<", "<< velMag<<", "<< rP.x<<", "<< rP.y<<", "<< rP.z<<", "<< rP.w<<endl;
			}
			fileNameFluidBoundaries<<ssFluidBoundaryParticles.str();
			fileNameFluidBoundaries.close();
		}
	posRadH.clear();
	velMasH.clear();
	rhoPresMuH.clear();
	rigidIdentifierH.clear();
	posRigidH.clear();
	posRigidCumulativeH.clear();
	velMassRigidH.clear();
	qH1.clear();
	omegaLRF_H.clear();
}

//*******************************************************************************************************************************
void PrintToFileDistribution(
		thrust::device_vector<int> & distributionD,
		real_ channelRadius,
		int numberOfSections,
		int tStep) {
	real_ dR = channelRadius / numberOfSections;
	int stepSaveDistribution = 1000;
	FILE *fileNameRadialDistribution;
	FILE *fileNameRadialDistribution_Normalized;
	if (tStep % stepSaveDistribution == 0) {
		if (tStep / stepSaveDistribution == 0) {
			fileNameRadialDistribution = fopen("radialDistribution.txt", "w");
			fprintf(fileNameRadialDistribution,"variables = \"r\", \"N\"\n");
			fileNameRadialDistribution_Normalized = fopen("radialDistributionNormalized.txt", "w");
			fprintf(fileNameRadialDistribution_Normalized,"variables = \"r\", \"N/r\"\n");
		} else {
			fileNameRadialDistribution = fopen("radialDistribution.txt", "a");
			fileNameRadialDistribution_Normalized = fopen("radialDistributionNormalized.txt", "a");
		}
		fprintf(fileNameRadialDistribution, "zone\n");
		fprintf(fileNameRadialDistribution_Normalized, "zone\n");
		for (int i = 0; i < distributionD.size(); i++) {
			real_ radialPos = dR * (i + 1);
			int distribution = distributionD[i];
			fprintf(fileNameRadialDistribution, "%f, %d\n", radialPos, distribution);
			fprintf(fileNameRadialDistribution_Normalized, "%f, %f\n", radialPos, distribution / radialPos);
		}
		fflush(fileNameRadialDistribution);
		fclose(fileNameRadialDistribution);
		fflush(fileNameRadialDistribution_Normalized);
		fclose(fileNameRadialDistribution_Normalized);
	}
}
